#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>

__global__ void insideTheUnitCircle ( const int n, const float *r, float *p ) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if ( i < n ) {
    p[i] = ( powf ( r[i], 2 ) + powf ( r[i+1], 2 ) =< 1 ) * 1.;
  }
}

int main ( int argc, char *argv[] ) {
  const int n = atoi ( argv[1] );
  const int nThr = atoi ( argv[2] );
  const int incxx = 1;

  int dev = 0;

  hipError_t err = hipSuccess;
  hipblasHandle_t cublasHandle = 0;
  hiprandGenerator_t gen;
  hipEvent_t start, stop;

  hipEventCreate ( &start );
  hipEventCreate ( &stop );
  hipEventRecord ( start, 0 );

  hipSetDevice ( dev );
  hipblasCreate ( &cublasHandle );

  hiprandCreateGenerator ( &gen, HIPRAND_RNG_PSEUDO_DEFAULT );
  hiprandSetPseudoRandomGeneratorSeed ( gen, 1234ULL );

  float *r, *p, res;
  hipMallocManaged ( ( void ** ) &r, 2 * n * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &p, n * sizeof ( float ) );

  dim3 block ( nThr );
  dim3 grid ( ( n + block.x - 1 ) / block.x );

  hiprandGenerateUniform ( gen, r, 2 * n );

  dim3 block ( nThr );
  dim3 grid ( ( n + block.x - 1 ) / block.x );

  insideTheUnitCircle <<< grid, block >>> ( n, r, p );

  hipblasSasum ( cublasHandle, n, p, incxx, &res );

  float elapsedTime;
  hipEventRecord ( stop, 0 );
  hipEventSynchronize ( stop );
  hipEventElapsedTime ( &elapsedTime, start, stop );

  printf ( "%1.8f\n", 4 * res / n );
  printf ( " Time to generate: %3.1f ms\n", elapsedTime );

  hipblasDestroy ( cublasHandle );
  hiprandDestroyGenerator ( gen );

  hipFree ( r );
  hipFree ( p );

  err = hipDeviceReset ();
  if ( err != hipSuccess ) {
    fprintf ( stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString ( err ) );
    exit ( EXIT_FAILURE );
  }

  return 0;
}
