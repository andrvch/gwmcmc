#ifndef _GWMCMCCUDA_CU_
#define _GWMCMCCUDA_CU_

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <fitsio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include "StrctrsAndFnctns.cuh"

int main ( int argc, char *argv[] ) {
  const int vrb = 1;

  Cupar cdp[1];
  cdp[0].dev = atoi ( argv[1] );

  initializeCuda ( cdp );

  if ( vrb ) {
    printf ( "\n" );
    printf ( " Device ID: %d\n", cdp[0].dev );
    printf ( " Device name: %s\n", cdp[0].prop.name );
    printf ( " Driver API: v%d \n", cdp[0].driverVersion[0] );
    printf ( " Runtime API: v%d \n", cdp[0].runtimeVersion[0] );
    printf ( "\n" );
  }

  Chain chn[1];
  chn[0].name = argv[2];
  chn[0].nwl = atoi ( argv[3] );
  chn[0].nst = atoi ( argv[4] );

  allocateChainForAutoCorr ( chn );

  simpleReadDataFloat ( chn[0].name, chn[0].chnFnctn );

  /*
  for ( int i = 0; i < chn[0].nwl*chn[0].nst; i++ ) {
    printf ( " %i ", i );
    printf ( " %2.4f ", chn[0].chnFnctn[i] );
    printf ( "\n" );
  }*/

  hipEventRecord ( cdp[0].start, 0 );

  atcrrltnfnctn ( cdp, chn );

  hipEventRecord ( cdp[0].stop, 0 );
  hipEventSynchronize ( cdp[0].stop );
  hipEventElapsedTime ( &chn[0].time, cdp[0].start, cdp[0].stop );

  if ( vrb ) {
    printf ( " Autocorrelation time window: %i\n", chn[0].mmm );
    printf ( " Autocorrelation time: %.8E\n", chn[0].atcTime );
    printf ( " Autocorrelation time threshold: %.8E\n", chn[0].nst / 5e1f );
    printf ( " Effective number of independent samples: %.8E\n", chn[0].nwl * chn[0].nst / chn[0].atcTime );
    printf ( " Time to compute acor time: %3.1f ms\n", chn[0].time );
    printf ( "\n" );
  }

  /* Write results to a file */
  printf ( " Write results to the host memory and clean up ... \n" );
  simpleWriteDataFloat ( "acrr.out", chn[0].nst, chn[0].atcrrFnctn );
  simpleWriteDataFloat ( "acrrcm.out", chn[0].nst, chn[0].cmSmAtCrrFnctn );

  destroyCuda ( cdp );
  freeChainForAutoCorr ( chn );

  // Reset the device and exit
  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application exits

  cdp[0].err = hipDeviceReset ();
  if ( cdp[0].err != hipSuccess ) {
    fprintf ( stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString ( cdp[0].err ) );
    exit ( EXIT_FAILURE );
  }

  printf ( " Done!\n" );

  return 0;
}

#endif // _GWMCMCCUDA_CU_
