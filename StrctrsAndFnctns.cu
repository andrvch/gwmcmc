#include "hip/hip_runtime.h"
#ifndef _STRCTRSANDFNCTNS_CU_
#define _STRCTRSANDFNCTNS_CU_

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <fitsio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include "StrctrsAndFnctns.cuh"

__host__ int SpecData ( Cuparam *cdp, const int verbose, Model *mdl, Spectrum *spc )
{
  float smOfNtcdChnnls = 0;
  for ( int i = 0; i < NSPCTR; i++ )
  {
    if ( verbose == 1 )
    {
      printf ( ".................................................................\n" );
      printf ( " Spectrum number  -- %i\n", i );
      printf ( " Spectrum table   -- %s\n", spc[i].srcTbl );
      printf ( " ARF table        -- %s\n", spc[i].arfTbl );
      printf ( " RMF table        -- %s\n", spc[i].rmfTbl );
      printf ( " Background table -- %s\n", spc[i].bckgrndTbl );
    }
    ReadFitsData ( verbose, spc[i].srcTbl, spc[i].arfTbl, spc[i].rmfTbl, spc[i].bckgrndTbl, spc[i].nmbrOfEnrgChnnls, spc[i].nmbrOfChnnls, spc[i].nmbrOfRmfVls, &spc[i].backscal_src, &spc[i].backscal_bkg, spc[i].srcCnts, spc[i].bckgrndCnts, spc[i].arfFctrs, spc[i].rmfVlsInCsc, spc[i].rmfIndxInCsc, spc[i].rmfPntrInCsc, spc[i].gdQltChnnls, spc[i].lwrChnnlBndrs, spc[i].hghrChnnlBndrs, spc[i].enrgChnnls );

    cdp[0].cusparseStat = hipsparseScsr2csc ( cdp[0].cusparseHandle, spc[i].nmbrOfEnrgChnnls, spc[i].nmbrOfChnnls, spc[i].nmbrOfRmfVls, spc[i].rmfVlsInCsc, spc[i].rmfPntrInCsc, spc[i].rmfIndxInCsc, spc[i].rmfVls, spc[i].rmfIndx, spc[i].rmfPntr, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO );
    if ( cdp[0].cusparseStat != HIPSPARSE_STATUS_SUCCESS ) { fprintf ( stderr, " CUSPARSE error: RMF transpose failed " ); return 1; }

    AssembleArrayOfNoticedChannels <<< Blocks ( spc[i].nmbrOfChnnls ), THRDSPERBLCK >>> ( spc[i].nmbrOfChnnls, spc[i].lwrNtcdEnrg, spc[i].hghrNtcdEnrg, spc[i].lwrChnnlBndrs, spc[i].hghrChnnlBndrs, spc[i].gdQltChnnls, spc[i].ntcdChnnls );
    cdp[0].cublasStat = hipblasSdot ( cdp[0].cublasHandle, spc[i].nmbrOfChnnls, spc[i].ntcdChnnls, INCXX, spc[i].ntcdChnnls, INCYY, &spc[i].smOfNtcdChnnls );
    if ( cdp[0].cublasStat != HIPBLAS_STATUS_SUCCESS ) { fprintf ( stderr, " CUBLAS error: channel summation failed " ); return 1; }
    hipDeviceSynchronize ( );
    smOfNtcdChnnls = smOfNtcdChnnls + spc[i].smOfNtcdChnnls;
    AssembleArrayOfPhotoelectricCrossections ( spc[i].nmbrOfEnrgChnnls, ATNMR, mdl[0].sgFlg, spc[i].enrgChnnls, mdl[0].atmcNmbrs, spc[i].crssctns );
    if ( verbose == 1 )
    {
      printf ( " Number of energy channels                = %i\n", spc[i].nmbrOfEnrgChnnls );
      printf ( " Number of instrument channels            = %i\n", spc[i].nmbrOfChnnls );
      printf ( " Number of nonzero elements of RMF matrix = %i\n", spc[i].nmbrOfRmfVls );
      printf ( " Exposure time                            = %.8E\n", spc[i].srcExptm );
      printf ( " Exposure time (background)               = %.8E\n", spc[i].bckgrndExptm );
      printf ( " Number of used instrument channels -- %4.0f\n", spc[i].smOfNtcdChnnls );
      printf ( " Backscale src -- %4.0f\n", spc[i].backscal_src );
      printf ( " Backscale bkg -- %4.0f\n", spc[i].backscal_bkg );
    }
  }
  if ( verbose == 1 )
  {
    printf ( ".................................................................\n" );
    printf ( " Total number of used instrument channels -- %4.0f\n", smOfNtcdChnnls );
    printf ( " Number of degrees of freedom -- %4.0f\n", smOfNtcdChnnls - NPRS );
  }
  return 0;
}

__host__ int SpecInfo ( const char *spcLst[NSPCTR], const int verbose, Spectrum *spc )
{
  for ( int i = 0; i < NSPCTR; i++ )
  {
    ReadFitsInfo ( spcLst[i], &spc[i].nmbrOfEnrgChnnls, &spc[i].nmbrOfChnnls, &spc[i].nmbrOfRmfVls, &spc[i].srcExptm, &spc[i].bckgrndExptm, spc[i].srcTbl, spc[i].arfTbl, spc[i].rmfTbl, spc[i].bckgrndTbl );
  }
  return 0;
}

__host__ int SpecAlloc ( Chain *chn, Spectrum *spc )
{
  for ( int i = 0; i < NSPCTR; i++ )
  {
    hipMallocManaged ( ( void ** ) &spc[i].rmfPntrInCsc, ( spc[i].nmbrOfEnrgChnnls + 1 ) * sizeof ( int ) );
    hipMallocManaged ( ( void ** ) &spc[i].rmfIndxInCsc, spc[i].nmbrOfRmfVls * sizeof ( int ) );
    hipMallocManaged ( ( void ** ) &spc[i].rmfPntr, ( spc[i].nmbrOfChnnls + 1 ) * sizeof ( int ) );
    hipMallocManaged ( ( void ** ) &spc[i].rmfIndx, spc[i].nmbrOfRmfVls * sizeof ( int ) );
    hipMallocManaged ( ( void ** ) &spc[i].rmfVlsInCsc, spc[i].nmbrOfRmfVls * sizeof ( float ) );
    hipMallocManaged ( ( void ** ) &spc[i].rmfVls, spc[i].nmbrOfRmfVls * sizeof ( float ) );
    hipMallocManaged ( ( void ** ) &spc[i].enrgChnnls, ( spc[i].nmbrOfEnrgChnnls + 1 ) * sizeof ( float ) );
    hipMallocManaged ( ( void ** ) &spc[i].arfFctrs, spc[i].nmbrOfEnrgChnnls * sizeof ( float ) );
    hipMallocManaged ( ( void ** ) &spc[i].srcCnts, spc[i].nmbrOfChnnls * sizeof ( float ) );
    hipMallocManaged ( ( void ** ) &spc[i].bckgrndCnts, spc[i].nmbrOfChnnls * sizeof ( float ) );
    hipMallocManaged ( ( void ** ) &spc[i].lwrChnnlBndrs, spc[i].nmbrOfChnnls * sizeof ( float ) );
    hipMallocManaged ( ( void ** ) &spc[i].hghrChnnlBndrs, spc[i].nmbrOfChnnls * sizeof ( float ) );
    hipMallocManaged ( ( void ** ) &spc[i].gdQltChnnls, spc[i].nmbrOfChnnls * sizeof ( float ) );
    hipMallocManaged ( ( void ** ) &spc[i].crssctns, spc[i].nmbrOfEnrgChnnls * ATNMR * sizeof ( float ) );
    hipMallocManaged ( ( void ** ) &spc[i].absrptnFctrs, spc[i].nmbrOfEnrgChnnls * chn[0].nmbrOfWlkrs * sizeof ( float ) );
    hipMallocManaged ( ( void ** ) &spc[i].mdlFlxs, spc[i].nmbrOfEnrgChnnls * chn[0].nmbrOfWlkrs * sizeof ( float ) );
    hipMallocManaged ( ( void ** ) &spc[i].nsa1Flxs, ( spc[i].nmbrOfEnrgChnnls + 1 ) * chn[0].nmbrOfWlkrs * sizeof ( float ) );
    hipMallocManaged ( ( void ** ) &spc[i].nsa2Flxs, ( spc[i].nmbrOfEnrgChnnls + 1 ) * chn[0].nmbrOfWlkrs * sizeof ( float ) );
    hipMallocManaged ( ( void ** ) &spc[i].flddMdlFlxs, spc[i].nmbrOfChnnls * chn[0].nmbrOfWlkrs * sizeof ( float ) );
    hipMallocManaged ( ( void ** ) &spc[i].ntcdChnnls, spc[i].nmbrOfChnnls * sizeof ( float ) );
    hipMallocManaged ( ( void ** ) &spc[i].chnnlSttstcs, spc[i].nmbrOfChnnls * chn[0].nmbrOfWlkrs * sizeof ( float ) );
    hipMallocManaged ( ( void ** ) &spc[i].tmsSttstcs, spc[i].nmbrOfPhtns * chn[0].nmbrOfWlkrs * sizeof ( float ) );
    hipMallocManaged ( ( void ** ) &spc[i].arrTms, spc[i].nmbrOfPhtns * sizeof ( float ) );
  }
  return 0;
}

__host__ int ToChain ( const int stpIndx, Chain *chn )
{
  WriteWalkersAndStatisticsToChain <<< Blocks ( chn[0].nmbrOfWlkrs ), THRDSPERBLCK >>> ( chn[0].nmbrOfWlkrs, stpIndx, chn[0].wlkrs, chn[0].sttstcs, chn[0].prrs, chn[0].chnOfWlkrs, chn[0].chnOfSttstcs, chn[0].chnOfPrrs );
  return 0;
}

__host__ int Update ( const int stpIndx, const int sbstIndx, Chain *chn )
{
  UpdateWalkers <<< Blocks ( chn[0].nmbrOfWlkrs / 2 ), THRDSPERBLCK >>> ( chn[0].nmbrOfWlkrs / 2, stpIndx, sbstIndx, chn[0].prpsdWlkrs, chn[0].prpsdSttstcs, chn[0].prpsdPrrs, chn[0].zRndmVls, chn[0].rndmVls, chn[0].wlkrs, chn[0].sttstcs, chn[0].prrs );
  return 0;
}

__host__ int Propose ( const int stpIndx, const int sbstIndx, Chain *chn )
{
  GenerateProposal <<< Blocks ( chn[0].nmbrOfWlkrs / 2 ), THRDSPERBLCK >>> ( chn[0].nmbrOfWlkrs / 2, stpIndx, sbstIndx, chn[0].wlkrs, chn[0].rndmVls, chn[0].zRndmVls, chn[0].prpsdWlkrs, chn[0].prpsdSttstcs );
  return 0;
}

__host__ int InitFromLast ( Chain *chn )
{
  InitializeWalkersAndStatisticsFromLastChain <<< Blocks ( chn[0].nmbrOfWlkrs ), THRDSPERBLCK >>> ( chn[0].nmbrOfWlkrs, chn[0].lstWlkrsAndSttstcs, chn[0].wlkrs, chn[0].sttstcs, chn[0].prrs );
  return 0;
}

__host__ int InitAtRandom ( Cuparam *cdp, Chain *chn )
{
  hiprandGenerateUniform ( cdp[0].curandGnrtr, chn[0].rndmVls, chn[0].nmbrOfWlkrs * NPRS );
  AssembleArrayOfRandomWalkers <<< Blocks ( chn[0].nmbrOfWlkrs ), THRDSPERBLCK >>> ( chn[0].nmbrOfWlkrs, chn[0].rndmVls, chn[0].rndmWlkr );
  InitializeWalkersAtRandom <<< Blocks ( chn[0].nmbrOfWlkrs ), THRDSPERBLCK >>> ( chn[0].nmbrOfWlkrs, chn[0].dlt, chn[0].strtngWlkr, chn[0].rndmWlkr, chn[0].wlkrs, chn[0].sttstcs );
  return 0;
}

__host__ int Stat ( const int nmbrOfWlkrs, Spectrum spec )
{
  dim3 dimBlock ( THRDSPERBLCK, THRDSPERBLCK );
  dim3 dimGrid = Grid ( spec.nmbrOfChnnls, nmbrOfWlkrs );
  AssembleArrayOfChannelStatistics <<< dimGrid, dimBlock >>> ( nmbrOfWlkrs, spec.nmbrOfChnnls, spec.srcExptm, spec.bckgrndExptm, spec.backscal_src, spec.backscal_bkg, spec.srcCnts, spec.bckgrndCnts, spec.flddMdlFlxs, spec.chnnlSttstcs );
  return 0;
}

__host__ int StatTimes ( const int nmbrOfWlkrs, const Walker *wlk, Spectrum spec )
{
  dim3 dimBlock ( THRDSPERBLCK, THRDSPERBLCK );
  dim3 dimGrid = Grid ( spec.nmbrOfPhtns, nmbrOfWlkrs );
  AssembleArrayOfTimesStatistic <<< dimGrid, dimBlock >>> ( nmbrOfWlkrs, spec.nmbrOfPhtns, spec.srcExptm, wlk, spec.arrTms, spec.tmsSttstcs );
  return 0;
}

__host__ int SumUpStat ( Cuparam *cdp, const float beta, const int nmbrOfWlkrs, float *sttstcs, const Spectrum spec )
{
  float alpha = ALPHA;
  cdp[0].cublasStat = hipblasSgemv ( cdp[0].cublasHandle, HIPBLAS_OP_T, spec.nmbrOfChnnls, nmbrOfWlkrs, &alpha, spec.chnnlSttstcs, spec.nmbrOfChnnls, spec.ntcdChnnls, INCXX, &beta, sttstcs, INCYY );
  if ( cdp[0].cublasStat != HIPBLAS_STATUS_SUCCESS ) { fprintf ( stderr, " CUBLAS error: Matrix-vector multiplication failed 0 " ); return 1; }
  return 0;
}

__host__ int FoldModel ( Cuparam *cdp, const int nmbrOfWlkrs, Spectrum spec )
{
  float alpha = ALPHA, beta = BETA;
  cdp[0].cusparseStat = hipsparseScsrmm ( cdp[0].cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, spec.nmbrOfChnnls, nmbrOfWlkrs, spec.nmbrOfEnrgChnnls, spec.nmbrOfRmfVls, &alpha, cdp[0].MatDescr, spec.rmfVls, spec.rmfPntr, spec.rmfIndx, spec.mdlFlxs, spec.nmbrOfEnrgChnnls, &beta, spec.flddMdlFlxs, spec.nmbrOfChnnls );
  if ( cdp[0].cusparseStat != HIPSPARSE_STATUS_SUCCESS ) { fprintf ( stderr, " CUSPARSE error: Matrix-matrix multiplication failed yes " ); return 1; }
  return 0;
}

__host__ void DestroyAllTheCudaStaff ( const Cuparam *cdp )
{
  hipsparseDestroy ( cdp[0].cusparseHandle );
  hipblasDestroy ( cdp[0].cublasHandle );
  hiprandDestroyGenerator ( cdp[0].curandGnrtr );
  hiprandDestroyGenerator ( cdp[0].curandGnrtrHst );
  hipEventDestroy ( cdp[0].start );
  hipEventDestroy ( cdp[0].stop );
  hipfftDestroy ( cdp[0].cufftPlan );
}

__host__ void FreeSpec ( const Spectrum *spc )
{
  for ( int i = 0; i < NSPCTR; i++ )
  {
    hipFree ( spc[i].rmfVlsInCsc );
    hipFree ( spc[i].rmfIndxInCsc );
    hipFree ( spc[i].rmfPntrInCsc );
    hipFree ( spc[i].rmfVls );
    hipFree ( spc[i].rmfIndx );
    hipFree ( spc[i].rmfPntr );
    hipFree ( spc[i].enrgChnnls );
    hipFree ( spc[i].arfFctrs );
    hipFree ( spc[i].srcCnts );
    hipFree ( spc[i].bckgrndCnts );
    hipFree ( spc[i].gdQltChnnls );
    hipFree ( spc[i].lwrChnnlBndrs );
    hipFree ( spc[i].hghrChnnlBndrs );
    hipFree ( spc[i].crssctns );
    hipFree ( spc[i].absrptnFctrs );
    hipFree ( spc[i].mdlFlxs );
    hipFree ( spc[i].nsa1Flxs );
    hipFree ( spc[i].nsa2Flxs );
    hipFree ( spc[i].flddMdlFlxs );
    hipFree ( spc[i].chnnlSttstcs );
    hipFree ( spc[i].ntcdChnnls );
    hipFree ( spc[i].tmsSttstcs );
    hipFree ( spc[i].arrTms );
  }
}

__host__ void FreeChain ( const Chain *chn )
{
  hipFree ( chn[0].wlkrs );
  hipFree ( chn[0].prpsdWlkrs );
  hipFree ( chn[0].chnOfWlkrs );
  hipFree ( chn[0].sttstcs );
  hipFree ( chn[0].prrs );
  hipFree ( chn[0].prpsdSttstcs );
  hipFree ( chn[0].prpsdPrrs );
  hipFree ( chn[0].zRndmVls );
  hipFree ( chn[0].chnOfSttstcs );
  hipFree ( chn[0].chnOfPrrs );
  hipFree ( chn[0].nhMd );
  hipFree ( chn[0].nhSg );
  hipFree ( chn[0].rndmVls );
  hipFree ( chn[0].rndmWlkr );
  hipFree ( chn[0].chnFnctn );
  hipFree ( chn[0].atCrrFnctn );
  hipFree ( chn[0].cmSmAtCrrFnctn );
  hipFree ( chn[0].lstWlkrsAndSttstcs );
}

__host__ void FreeModel ( const Model *mdl )
{
  hipFree ( mdl[0].atmcNmbrs );
  hipFree ( mdl[0].abndncs );
  hipFree ( mdl[0].RedData );
  hipFree ( mdl[0].Dist );
  hipFree ( mdl[0].EBV );
  hipFree ( mdl[0].errDist );
  hipFree ( mdl[0].errEBV );
  hipFree ( mdl[0].RedData1 );
  hipFree ( mdl[0].Dist1 );
  hipFree ( mdl[0].EBV1 );
  hipFree ( mdl[0].nsaDt );
  hipFree ( mdl[0].nsaT );
  hipFree ( mdl[0].nsaE );
  hipFree ( mdl[0].nsaFlxs );
  hipFree ( mdl[0].nsmaxgDt );
  hipFree ( mdl[0].nsmaxgT );
  hipFree ( mdl[0].nsmaxgE );
  hipFree ( mdl[0].nsmaxgFlxs );
}

__host__ int InitializeCuda ( Cuparam *cdp )
{
  /* cuda runtime version */
  hipRuntimeGetVersion ( cdp[0].runtimeVersion );
  hipDriverGetVersion ( cdp[0].driverVersion );
  /* Set and enquire about cuda device */
  hipSetDevice ( cdp[0].dev );
  hipGetDevice ( &cdp[0].dev );
  hipGetDeviceProperties ( &cdp[0].prop, cdp[0].dev );
  /* cuSparse related things */
  cdp[0].cusparseStat = hipsparseCreate ( &cdp[0].cusparseHandle );
  if ( cdp[0].cusparseStat != HIPSPARSE_STATUS_SUCCESS ) { fprintf ( stderr, " CUSPARSE error: Creation of cuSparse context failed " ); return 1; }
  cdp[0].cusparseStat = hipsparseCreateMatDescr ( &cdp[0].MatDescr );
  if ( cdp[0].cusparseStat != HIPSPARSE_STATUS_SUCCESS ) { fprintf ( stderr, " CUSPARSE error: Creation of matrix descriptor failed " ); return 1; }
  cdp[0].cusparseStat = hipsparseSetMatType ( cdp[0].MatDescr, HIPSPARSE_MATRIX_TYPE_GENERAL );
  if ( cdp[0].cusparseStat != HIPSPARSE_STATUS_SUCCESS ) { fprintf ( stderr, " CUSPARSE error: Setting matrix type to general failed " ); return 1; }
  cdp[0].cusparseStat = hipsparseSetMatIndexBase ( cdp[0].MatDescr, HIPSPARSE_INDEX_BASE_ZERO );
  if ( cdp[0].cusparseStat != HIPSPARSE_STATUS_SUCCESS ) { fprintf ( stderr, " CUSPARSE error: Setting to base zero index failed " ); return 1; }
  /* cuBlas related things */
  cdp[0].cublasStat = hipblasCreate ( &cdp[0].cublasHandle );
  if ( cdp[0].cublasStat != HIPBLAS_STATUS_SUCCESS ) { fprintf ( stderr, " CUBLAS error: Creation of cuBlas context failed " ); return 1; }
  /* cuRand related things */
  hiprandCreateGenerator ( &cdp[0].curandGnrtr, HIPRAND_RNG_PSEUDO_DEFAULT );
  hiprandCreateGeneratorHost ( &cdp[0].curandGnrtrHst, HIPRAND_RNG_PSEUDO_DEFAULT );
  hiprandSetPseudoRandomGeneratorSeed ( cdp[0].curandGnrtr, 1234ULL );
  hiprandSetPseudoRandomGeneratorSeed ( cdp[0].curandGnrtrHst, 1234ULL );
  /* cuFfft related things */
  hipEventCreate ( &cdp[0].start );
  hipEventCreate ( &cdp[0].stop );
  printf ( "\n" );
  printf ( ".................................................................\n" );
  printf ( " CUDA device ID: %d\n", cdp[0].dev );
  printf ( " CUDA device Name: %s\n", cdp[0].prop.name );
  printf ( " Driver API: v%d \n", cdp[0].driverVersion[0] );
  printf ( " Runtime API: v%d \n", cdp[0].runtimeVersion[0] );
  return 0;
}

__host__ int InitializeModel ( Model *mdl )
{
  hipMallocManaged ( ( void ** ) &mdl[0].atmcNmbrs, ATNMR * sizeof ( int ) );
  hipMallocManaged ( ( void ** ) &mdl[0].abndncs, ( NELMS + 1 ) * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &mdl[0].RedData, mdl[0].nmbrOfDistBins * mdl[0].numRedCol * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &mdl[0].Dist, mdl[0].nmbrOfDistBins * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &mdl[0].EBV, mdl[0].nmbrOfDistBins * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &mdl[0].RedData1, mdl[0].nmbrOfDistBins1 * mdl[0].numRedCol1 * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &mdl[0].Dist1, mdl[0].nmbrOfDistBins1 * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &mdl[0].EBV1, mdl[0].nmbrOfDistBins1 * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &mdl[0].errDist, mdl[0].nmbrOfDistBins * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &mdl[0].errEBV, mdl[0].nmbrOfDistBins * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &mdl[0].nsaDt, ( mdl[0].numNsaE + 1 ) * ( mdl[0].numNsaT + 1 ) * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &mdl[0].nsaE, mdl[0].numNsaE * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &mdl[0].nsaT, mdl[0].numNsaT * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &mdl[0].nsaFlxs, mdl[0].numNsaE * mdl[0].numNsaT * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &mdl[0].nsmaxgDt, ( mdl[0].numNsaE + 1 ) * ( mdl[0].numNsaT + 1 ) * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &mdl[0].nsmaxgE, mdl[0].numNsaE * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &mdl[0].nsmaxgT, mdl[0].numNsaT * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &mdl[0].nsmaxgFlxs, mdl[0].numNsaE * mdl[0].numNsaT * sizeof ( float ) );
  for ( int i = 0; i < ATNMR; i++ ) { mdl[0].atmcNmbrs[i] = mdl[0].atNm[i]; }
  SimpleReadDataFloat ( mdl[0].abndncsFl, mdl[0].abndncs );
  SimpleReadReddenningData ( mdl[0].rddnngFl, mdl[0].nmbrOfDistBins, mdl[0].RedData, mdl[0].Dist, mdl[0].EBV, mdl[0].errDist, mdl[0].errEBV );
  SimpleReadReddenningDataNoErrors ( mdl[0].rddnngFl1, mdl[0].nmbrOfDistBins1, mdl[0].RedData1, mdl[0].Dist1, mdl[0].EBV1 );
  SimpleReadNsaTable ( mdl[0].nsaFl, mdl[0].numNsaE, mdl[0].numNsaT, mdl[0].nsaDt, mdl[0].nsaT, mdl[0].nsaE, mdl[0].nsaFlxs );
  SimpleReadNsmaxgTable ( mdl[0].nsmaxgFl, mdl[0].numNsmaxgE, mdl[0].numNsmaxgT, mdl[0].nsmaxgDt, mdl[0].nsmaxgT, mdl[0].nsmaxgE, mdl[0].nsmaxgFlxs );
  return 0;
}

__host__ int InitializeChain ( Cuparam *cdp, const float *phbsPwrlwInt, Chain *chn )
{
  int prmtrIndx = 0;
  chn[0].nmbrOfRndmVls = 3 * chn[0].nmbrOfWlkrs / 2 * chn[0].nmbrOfStps;
  hipMallocManaged ( ( void ** ) &chn[0].wlkrs, chn[0].nmbrOfWlkrs * sizeof ( Walker ) );
  hipMallocManaged ( ( void ** ) &chn[0].prpsdWlkrs, chn[0].nmbrOfWlkrs / 2 * sizeof ( Walker ) );
  hipMallocManaged ( ( void ** ) &chn[0].chnOfWlkrs, chn[0].nmbrOfWlkrs * chn[0].nmbrOfStps * sizeof ( Walker ) );
  hipMallocManaged ( ( void ** ) &chn[0].sttstcs, chn[0].nmbrOfWlkrs * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &chn[0].prrs, chn[0].nmbrOfWlkrs * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &chn[0].prpsdSttstcs, chn[0].nmbrOfWlkrs / 2 * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &chn[0].prpsdPrrs, chn[0].nmbrOfWlkrs / 2 * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &chn[0].chnOfSttstcs, chn[0].nmbrOfWlkrs * chn[0].nmbrOfStps * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &chn[0].chnOfPrrs, chn[0].nmbrOfWlkrs * chn[0].nmbrOfStps * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &chn[0].zRndmVls, chn[0].nmbrOfWlkrs / 2 * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &chn[0].nhMd, chn[0].nmbrOfWlkrs * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &chn[0].nhSg, chn[0].nmbrOfWlkrs * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &chn[0].rndmVls, chn[0].nmbrOfRndmVls * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &chn[0].rndmWlkr, chn[0].nmbrOfWlkrs * sizeof ( Walker ) );
  hipMallocManaged ( ( void ** ) &chn[0].chnFnctn, chn[0].nmbrOfStps * chn[0].nmbrOfWlkrs * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &chn[0].atCrrFnctn, chn[0].nmbrOfStps * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &chn[0].cmSmAtCrrFnctn, chn[0].nmbrOfStps * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &chn[0].lstWlkrsAndSttstcs, ( NPRS + 2 ) * chn[0].nmbrOfWlkrs * sizeof ( float ) );
  if ( chn[0].thrdIndx > 0 )
  {
    ReadLastPositionOfWalkersFromFile ( chn[0].thrdNm, chn[0].thrdIndx-1, chn[0].nmbrOfWlkrs, chn[0].lstWlkrsAndSttstcs );
  }
  else if ( chn[0].thrdIndx == 0 )
  {
    for ( int i = 0; i < NPRS; i++ )
    {
      chn[0].strtngWlkr.par[i] = phbsPwrlwInt[i];
    }
    //hiprandGenerateUniform ( cdp[0].curandGnrtrHst, chn[0].rndmVls, ATNMR - 1 );
    //prmtrIndx = NHINDX + 1;
    //while ( prmtrIndx < NPRS )
    //{
    //  chn[0].strtngWlkr.par[prmtrIndx] = chn[0].dlt * ( 1 - 2 * chn[0].rndmVls[prmtrIndx-3] );
    //  prmtrIndx += 1;
    //}
    printf ( ".................................................................\n" );
    printf ( " Initial parameters -- " );
    prmtrIndx = 0;
    while ( prmtrIndx < NPRS )
    {
      printf ( " %2.2f ", chn[0].strtngWlkr.par[prmtrIndx] );
      prmtrIndx += 1;
    }
    printf ( "\n" );
    if ( not PriorCondition ( chn[0].strtngWlkr ) ) { printf ( " !!!Initial walker unsatisfy prior conditions!!!\n" ); }
  }
  return 0;
}

/* Functions: */
__host__ int Blocks ( const int n )
{
  int blcksPerThrd;
  blcksPerThrd = ( n + THRDSPERBLCK - 1 ) / THRDSPERBLCK;
  return blcksPerThrd;
}

__host__ dim3 Grid ( const int n, const int m )
{
  dim3 dimGrid ( ( n + THRDSPERBLCK - 1 ) / THRDSPERBLCK, ( m + THRDSPERBLCK - 1 ) / THRDSPERBLCK );
  return dimGrid;
}

__host__ __device__ Walker AddWalkers ( Walker a, Walker b )
{
  Walker c;
  for ( int i = 0; i < NPRS; i++ ) { c.par[i] = a.par[i] + b.par[i]; }
  return c;
}

__host__ __device__ float SumOfComponents ( const Walker wlkr )
{
  float mag = 0;
  for ( int i = FIRSTBIN; i < NPRS; i++ )
  {
    mag += wlkr.par[i];
  }
  return mag;
}

__host__ __device__ Walker ScaleWalker ( Walker a, float s )
{
  Walker c;
  for ( int i = 0; i < NPRS; i++ ) { c.par[i] = s * a.par[i]; }
  return c;
}

__host__ __device__ Complex AddComplex ( Complex a, Complex b )
{
  Complex c;
  c.x = a.x + b.x;
  c.y = a.y + b.y;
  return c;
}

__host__ __device__ Complex ScaleComplex ( Complex a, float s )
{
  Complex c;
  c.x = s * a.x;
  c.y = s * a.y;
  return c;
}

__host__ __device__ Complex MultiplyComplex ( Complex a, Complex b )
{
  Complex c;
  c.x = a.x * b.x - a.y * b.y;
  c.y = a.x * b.y + a.y * b.x;
  return c;
}

__host__ __device__ Complex ConjugateComplex ( Complex a )
{
  Complex c;
  c.x = a.x;
  c.y = - a.y;
  return c;
}

__host__ __device__ float GaussianAbsorption ( const float lcen, const float lwidth, const float ldepth, const float enrgHghr )
{
  float fctr;
  float depth = powf ( 10., ldepth );
  float width = powf ( 10., lwidth );
  float cen = powf ( 10., lcen );
  fctr = expf ( - depth / sqrtf ( 2 * PIPI ) / width * expf ( - 0.5 * powf ( ( enrgHghr - cen ) / width , 2. ) ) );
  return fctr;
}

__host__ __device__ float PowerLaw ( const float phtnIndx, const float nrmlztn, const float enrgLwr, const float enrgHghr )
{
  float flx;
  if ( fabsf ( 1 - phtnIndx ) > TLR )
  {
    flx = powf ( 10, nrmlztn ) * ( powf ( enrgHghr, 1 - phtnIndx ) - powf ( enrgLwr, 1 - phtnIndx ) ) / ( 1 - phtnIndx );
  }
  else
  {
    flx = powf ( 10, nrmlztn ) * ( logf ( enrgHghr ) - logf ( enrgLwr ) );
  }
  return flx;
}

__host__ __device__ float IntegrateNsa ( const float flx1, const float flx2, const float en1, const float en2 )
{
  float flx;
  flx = 0.5 * ( flx1 + flx2 ) * ( en2 - en1 );
  return flx;
}

__host__ __device__ float IntegrateNsmax ( const float flx1, const float flx2, const float en1, const float en2 )
{
  float flx;
  float gr = sqrtf ( 1. - 2.952 * MNS / RNS );
  flx = gr * powf ( 10, 26.1787440 ) * 0.5 * ( flx1 / en1 + flx2 / en2 ) * ( en2 - en1 );
  return flx;
}

__host__ __device__ float BlackBody ( const float kT, const float logRtD, const float enrgLwr, const float enrgHghr )
{
  float t, anorm, elow, x, tinv, anormh, alow, ehi, ahi, flx;
  t = kT;
  tinv = 1. / t;
  anorm = 1.0344e-3f * 1e8f * powf ( 10, 2 * logRtD ) ;
  anormh = 0.5 * anorm;
  elow = enrgLwr;
  x = elow * tinv;
  if ( x <= 1.0e-4f )
  {
    alow = elow * t;
  }
  else if ( x > 60.0 )
  {
    flx = 0;
    return flx;
  }
  else
  {
    alow = elow * elow / ( expf ( x ) - 1.0e0f );
  }
  ehi = enrgHghr;
  x = ehi * tinv;
  if ( x <= 1.0e-4f )
  {
    ahi = ehi * t;
  }
  else if ( x > 60.0 )
  {
    flx = 0;
    return flx;
  }
  else
  {
    ahi = ehi * ehi / ( expf ( x ) - 1.0e0f );
  }
  flx = anormh * ( alow + ahi ) * ( ehi - elow );
  return flx;
}

__host__ __device__ float Poisson ( const float scnts, const float mdl, const float ts )
{
  float sttstc = 0;
  if ( scnts != 0 && ts * mdl >= TLR )
  {
    sttstc = ts * mdl - scnts * logf ( ts * mdl ) - scnts * ( 1 - logf ( scnts ) );
  }
  else if ( scnts != 0 && ts * mdl < TLR )
  {
    sttstc = TLR - scnts * logf ( TLR ) - scnts * ( 1 - logf ( scnts ) );
  }
  else
  {
    sttstc = ts * mdl;
  }
  sttstc = 2 * sttstc;
  return sttstc;
}

__host__ __device__ float GregoryLoredo ( const float tms, const Walker wlkr, const float Ttot, const int N )
{
    float sttstc = 0, f, phi, jt, jtFr, jtInt, jtJt, A;
    f = wlkr.par[0] * 1.E-6 + F0;
    phi = wlkr.par[1];
    jt = 1 + ( NTBINS / ( 2 * PI ) ) * fmodf ( 2 * PI * f * tms + phi, 2 * PI );
    jtFr = modff( jt, &jtInt );
    jtJt = jt - jtFr;
    int jIndx = llroundf ( jtJt );
    A = SumOfComponents ( wlkr ) / NTBINS;
    sttstc = logf ( NTBINS * A ) - A * Ttot / N + logf ( wlkr.par[jIndx] / NTBINS / A );
    return sttstc;
}

__host__ __device__ float PoissonWithBackground ( const float scnts, const float bcnts, const float mdl, const float ts, const float tb, const float backscal_src, const float backscal_bkg )
{
  float sttstc = 0, d, f;
  float scls = 1;
  float sclb = backscal_bkg / backscal_src;
  d = sqrtf ( powf ( ( ts * scls + tb * sclb ) * mdl - scnts - bcnts, 2. ) + 4 * ( ts * scls + tb * sclb ) * bcnts * mdl );
  f = ( scnts + bcnts - ( ts * scls + tb * sclb ) * mdl + d ) / 2 / ( ts * scls + tb * sclb );
  if ( scnts != 0 && bcnts != 0 )
  {
    sttstc = ts * mdl + ts * scls * f  + tb * sclb * f - scnts * logf ( ts * mdl + ts * scls * f ) - bcnts * logf ( tb * sclb * f ) - scnts * ( 1 - logf ( scnts ) ) - bcnts * ( 1 - logf ( bcnts ) );
  }
  else if ( scnts != 0 && bcnts == 0 && mdl >= scnts / ( ts * scls + tb * sclb ) )
  {
    sttstc = ts * mdl - scnts * logf ( ts * mdl ) - scnts * ( 1 - logf ( scnts ) );
  }
  else if ( scnts != 0 && bcnts == 0 && mdl < scnts / ( ts * scls + tb * sclb ) )
  {
    sttstc = ts * ( 1 - scls ) * mdl - tb * sclb * mdl - scnts * logf ( ts * ( 1 - scls ) * mdl + ts * scls * scnts / ( ts * scls + tb * sclb ) ) + scnts * logf ( scnts );
  }
  else if ( scnts == 0 && bcnts != 0 )
  {
    sttstc = ts * mdl - bcnts * logf ( tb * sclb / ( ts * scls + tb * sclb ) );
  }
  else if ( scnts == 0 && bcnts == 0 )
  {
    sttstc = ts * mdl;
  }
  sttstc = 2 * sttstc;
  return sttstc;
}

__host__ __device__ int FindElementIndex ( const float *xx, const int n, const float x )
{
  int ju, jm, jl, jres;
  jl = 0;
  ju = n;
  while ( ju - jl > 1 )
  {
    jm = floorf ( 0.5 * ( ju + jl ) );
    if ( x >= xx[jm] ) { jl = jm; } else { ju = jm; }
  }
  jres = jl;
  if ( x == xx[0] ) jres = 0;
  if ( x >= xx[n-1] ) jres = n - 1;
  return jres;
}

__host__ void AssembleArrayOfPhotoelectricCrossections ( const int nmbrOfEnrgChnnls, const int nmbrOfElmnts, int sgFlag, float *enrgChnnls, int *atmcNmbrs, float *crssctns )
{
  int status = 0, versn = sgFlag, indx;
  for ( int i = 0; i < nmbrOfEnrgChnnls; i++ )
  {
    for ( int j = 0; j < nmbrOfElmnts; j++ )
    {
      indx = j + i * nmbrOfElmnts;
      crssctns[indx] = photo_ ( &enrgChnnls[i], &enrgChnnls[i+1], &atmcNmbrs[j], &versn, &status );
    }
  }
}

__host__ void ReadLastPositionOfWalkersFromFile ( const char *thrdNm, const int indx, const int nmbrOfWlkrs, float *lstChn )
{
  FILE *flPntr;
  char flNm[FLEN_CARD];
  float value;
  int i = 0, k = 0, j;
  snprintf ( flNm, sizeof ( flNm ), "%s%i%s", thrdNm, indx, ".chain" );
  flPntr = fopen ( flNm, "r" );
  while ( fscanf ( flPntr, "%e", &value ) == 1 )
  {
    i += 1;
  }
  fclose ( flPntr );
  flPntr = fopen ( flNm, "r" );
  while ( fscanf ( flPntr, "%e", &value ) == 1 )
  {
    if ( k >= i - nmbrOfWlkrs * ( NPRS + 2 ) )
    {
      j = k - ( i - nmbrOfWlkrs * ( NPRS + 2 ) );
      lstChn[j] = value;
    }
    k += 1;
  }
  fclose ( flPntr );
}

__host__ void WriteChainToFile ( const char *thrdNm, const int indx, const int nmbrOfWlkrs, const int nmbrOfStps, const Walker *chnOfWlkrs, const float *chnOfSttstcs, const float *chnOfPrrs )
{
  FILE *flPntr;
  char flNm[FLEN_CARD];
  int ttlChnIndx, stpIndx, wlkrIndx, prmtrIndx;
  snprintf ( flNm, sizeof ( flNm ), "%s%i%s", thrdNm, indx, ".chain" );
  flPntr = fopen ( flNm, "w" );
  stpIndx = 0;
  while ( stpIndx < nmbrOfStps )
  {
    wlkrIndx = 0;
    while ( wlkrIndx < nmbrOfWlkrs )
    {
      ttlChnIndx = wlkrIndx + stpIndx * nmbrOfWlkrs;
      prmtrIndx = 0;
      while ( prmtrIndx < NPRS )
      {
        fprintf ( flPntr, " %.8E ", chnOfWlkrs[ttlChnIndx].par[prmtrIndx] );
        prmtrIndx += 1;
      }
      fprintf ( flPntr, " %.8E ", chnOfSttstcs[ttlChnIndx] );
      prmtrIndx += 1;
      fprintf ( flPntr, " %.8E\n", chnOfPrrs[ttlChnIndx] );
      wlkrIndx += 1;
    }
    stpIndx += 1;
  }
  fclose ( flPntr );
}

__host__ void SimpleReadNsaTable ( const char *flNm, const int numEn, const int numTe, float *data, float *Te, float *En, float *fluxes )
{
  FILE *flPntr;
  float value;
  int i = 0;
  flPntr = fopen ( flNm, "r" );
  while ( fscanf ( flPntr, "%e", &value ) == 1 )
  {
    data[i] = value;
    i += 1;
  }
  for (int j = 0; j < numEn; j++)
  {
    En[j] = log10f ( data[(j+1)*(numTe+1)] );
  }
  for (int j = 0; j < numTe; j++)
  {
    Te[j] = data[j+1];
  }
  for (int j = 0; j < numEn; j++)
  {
    for (int i = 0; i < numTe; i++)
    {
      fluxes[j+i*numEn] = log10f ( data[(i+1)+(j+1)*(numTe+1)] );
    }
  }
  fclose ( flPntr );
}

__host__ void SimpleReadNsmaxgTable ( const char *flNm, const int numEn, const int numTe, float *data, float *Te, float *En, float *fluxes )
{
  FILE *flPntr;
  float value;
  int i = 0;
  flPntr = fopen ( flNm, "r" );
  while ( fscanf ( flPntr, "%e", &value ) == 1 )
  {
    data[i] = value;
    i += 1;
  }
  //numTe = (int*)data[0];
  for (int j = 0; j < numTe; j++)
  {
    Te[j] = data[1+j];
  }
  //numEn = (int*)data[17];
  for (int j = 0; j < numEn; j++)
  {
    En[j] = log10f ( data[18+j] );
  }
  for (int i = 0; i < numTe; i++)
  {
    for (int j = 0; j < numEn; j++)
    {
      fluxes[j+i*numEn] = log10f ( data[(18+numEn)+j+i*numEn] );
    }
  }
  fclose ( flPntr );
}

__host__ void SimpleReadReddenningData ( const char *flNm, const int numDist, float *data, float *Dist, float *EBV, float *errDist, float *errEBV )
{
  FILE *flPntr;
  float value;
  int i = 0;
  flPntr = fopen ( flNm, "r" );
  while ( fscanf (flPntr, "%e", &value ) == 1 )
  {
    data[i] = value;
    i += 1;
  }
  for ( int j = 0; j < numDist; j++ )
  {
    Dist[j] = log10f ( data[5*j] );
    EBV[j] = log10f ( data[5*j+1] );
    errDist[j] = log10f ( data[5*j+2] );
    errEBV[j] = log10f ( data[5*j+3] );
  }
  fclose ( flPntr );
}

__host__ void SimpleReadReddenningDataNoErrors ( const char *flNm, const int numDist, float *data, float *Dist, float *EBV )
{
  FILE *flPntr;
  float value;
  int i = 0;
  flPntr = fopen ( flNm, "r" );
  while ( fscanf (flPntr, "%e", &value ) == 1 )
  {
    data[i] = value;
    i += 1;
  }
  for ( int j = 0; j < numDist; j++ )
  {
    Dist[j] = log10f ( data[2*j] * 1000. );
    EBV[j] = log10f ( data[2*j+1] );
  }
  fclose ( flPntr );
}

__host__ void SimpleReadDataFloat ( const char *flNm, float *data )
{
  FILE *flPntr;
  float value;
  int i = 0;
  flPntr = fopen ( flNm, "r" );
  while ( fscanf ( flPntr, "%e", &value ) == 1 )
  {
    data[i] = value;
    i += 1;
  }
  fclose ( flPntr );
}

__host__ void SimpleReadDataInt ( const char *flNm, int *data )
{
  FILE *flPntr;
  int value;
  int i = 0;
  flPntr = fopen ( flNm, "r" );
  while ( fscanf ( flPntr, "%i", &value ) == 1 )
  {
    data[i] = value;
    i += 1;
  }
  fclose ( flPntr );
}

__host__ void SimpleWriteDataFloat ( const char *flNm, const int nmbrOfStps, const float *chn )
{
  FILE *flPntr;
  flPntr = fopen ( flNm, "w" );
  for ( int i = 0; i < nmbrOfStps; i++ )
  {
    fprintf ( flPntr, " %.8E\n", chn[i] );
  }
  fclose ( flPntr );
}

__host__ void SimpleWriteDataFloat2D ( const char *flNm, const int nmbrOfStps, const int nmbrOfWlkrs, const float *chn )
{
  FILE *flPntr;
  flPntr = fopen ( flNm, "w" );
  for ( int j = 0; j < nmbrOfStps; j++ )
  {
    for ( int i = 0; i < nmbrOfWlkrs; i++ )
    {
      fprintf ( flPntr, " %.8E ", chn[i+j*nmbrOfWlkrs] );
    }
    fprintf ( flPntr,  "\n" );
  }
  fclose ( flPntr );
}

__host__ void AutocorrelationFunctionAveraged ( hipfftResult_t cufftRes, hipblasStatus_t cublasStat, hipblasHandle_t cublasHandle, hipfftHandle cufftPlan, const int nmbrOfStps, const int nmbrOfWlkrs, const float *chnFnctn, float *atcrrFnctn )
{
  int incxx = INCXX, incyy = INCYY;
  float alpha = ALPHA, beta = BETA;
  int thrdsPerBlck = THRDSPERBLCK;
  dim3 dimBlock ( thrdsPerBlck, thrdsPerBlck );
  int blcksPerThrd_0 = ( nmbrOfStps + thrdsPerBlck - 1 ) / thrdsPerBlck;
  int blcksPerThrd_1 = ( nmbrOfWlkrs + thrdsPerBlck - 1 ) / thrdsPerBlck;
  dim3 dimGrid_0 ( ( nmbrOfWlkrs + thrdsPerBlck - 1 ) / thrdsPerBlck, ( nmbrOfStps + thrdsPerBlck - 1 ) / thrdsPerBlck );
  dim3 dimGrid_1 ( ( nmbrOfStps + thrdsPerBlck - 1 ) / thrdsPerBlck, ( nmbrOfWlkrs + thrdsPerBlck - 1 ) / thrdsPerBlck );
  dim3 dimGrid_2 ( ( nmbrOfStps + thrdsPerBlck - 1 ) / thrdsPerBlck, ( nmbrOfStps + thrdsPerBlck - 1 ) / thrdsPerBlck );
  float *stps, *smOfChn, *cntrlChnFnctn, *wlkrs, *cmSmMtrx;
  hipfftComplex *ftOfChn;
  hipMallocManaged ( ( void ** ) &stps, nmbrOfStps * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &smOfChn, nmbrOfWlkrs * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &cntrlChnFnctn, nmbrOfStps * nmbrOfWlkrs * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &wlkrs, nmbrOfWlkrs * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &ftOfChn, nmbrOfStps * nmbrOfWlkrs * sizeof ( hipfftComplex ) );
  hipMallocManaged ( ( void ** ) &cmSmMtrx, nmbrOfStps * nmbrOfStps * sizeof ( float ) );
  ReturnConstantArray <<< blcksPerThrd_0, thrdsPerBlck >>> ( nmbrOfStps, alpha / nmbrOfStps, stps );
  cublasStat = hipblasSgemv ( cublasHandle, HIPBLAS_OP_N, nmbrOfWlkrs, nmbrOfStps, &alpha, chnFnctn, nmbrOfWlkrs, stps, incxx, &beta, smOfChn, incyy );
  if ( cublasStat != HIPBLAS_STATUS_SUCCESS ) { fprintf ( stderr, " CUBLAS error: " ); }
  ReturnCentralChainFunction <<< dimGrid_0, dimBlock >>> ( nmbrOfStps, nmbrOfWlkrs, smOfChn, chnFnctn, cntrlChnFnctn );
  ReturnChainFunctionTest <<< dimGrid_0, dimBlock >>> ( nmbrOfStps, nmbrOfWlkrs, 0, cntrlChnFnctn, ftOfChn );
  cufftRes = hipfftExecC2C ( cufftPlan, ( hipfftComplex * ) ftOfChn, ( hipfftComplex * ) ftOfChn, HIPFFT_FORWARD );
  if ( cufftRes != HIPFFT_SUCCESS ) { fprintf ( stderr, "CUFFT error:" ); }
  ComplexPointwiseMultiplyByConjugateAndScale <<< dimGrid_1, dimBlock >>> ( nmbrOfStps, nmbrOfWlkrs, alpha / nmbrOfStps, ftOfChn );
  cufftRes = hipfftExecC2C ( cufftPlan, ( hipfftComplex * ) ftOfChn, ( hipfftComplex * ) ftOfChn, HIPFFT_BACKWARD );
  if ( cufftRes != HIPFFT_SUCCESS ) { fprintf ( stderr, "CUFFT error: " ); }
  ReturnChainFunctionTest <<< dimGrid_0, dimBlock >>> ( nmbrOfStps, nmbrOfWlkrs, 1, cntrlChnFnctn, ftOfChn );
  ReturnConstantArray <<< blcksPerThrd_1, thrdsPerBlck >>> ( nmbrOfWlkrs, alpha / nmbrOfWlkrs, wlkrs );
  cublasStat = hipblasSgemv ( cublasHandle, HIPBLAS_OP_T, nmbrOfWlkrs, nmbrOfStps, &alpha, cntrlChnFnctn, nmbrOfWlkrs, wlkrs, incxx, &beta, atcrrFnctn, incyy );
  if ( cublasStat != HIPBLAS_STATUS_SUCCESS ) { fprintf ( stderr, " CUBLAS error: " ); }
  NormalizeChain <<< blcksPerThrd_0, thrdsPerBlck >>> ( nmbrOfStps, atcrrFnctn );
  //MakeMatrix <<< dimGrid_2, dimBlock >>> ( nmbrOfStps, atcrrFnctn, cmSmMtrx );
  //ReturnConstantArray <<< blcksPerThrd_0, thrdsPerBlck >>> ( nmbrOfStps, alpha, stps );
  //cublasStat = hipblasSgemv ( cublasHandle, HIPBLAS_OP_T, nmbrOfStps, nmbrOfStps, &alpha, cmSmMtrx, nmbrOfStps, stps, incxx, &beta, cmSmAtcrrFnctn, incyy );
  //if ( cublasStat != HIPBLAS_STATUS_SUCCESS ) { fprintf ( stderr, " CUBLAS error: " ); }
  hipFree ( stps );
  hipFree ( smOfChn );
  hipFree ( cntrlChnFnctn );
  hipFree ( wlkrs );
  hipFree ( ftOfChn );
  hipFree ( cmSmMtrx );
}

__host__ void CumulativeSumOfAutocorrelationFunction ( const int nmbrOfStps, const float *chn, float *cmSmChn )
{
  float sum = 0;
  for ( int i = 0; i < nmbrOfStps; i++ )
  {
    sum = sum + chn[i];
    cmSmChn[i] = sum;
  }
}

__host__ int ChooseWindow ( const int nmbrOfStps, const float c, const float *cmSmChn )
{
  int m = 0;
  while ( ( m < c * ( 2 * cmSmChn[m] - 1e0f ) ) && ( m < nmbrOfStps )  )
  {
    m += 1;
  }
  return m;
}

/* Kernels: */
__global__ void AssembleArrayOfRandomWalkers ( const int nmbrOfWlkrs, const float *rndmVls, Walker *rndmWlkr )
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if ( i < nmbrOfWlkrs )
  {
    for ( int p = 0; p < NPRS; p++ )
    {
      rndmWlkr[i].par[p] = rndmVls[p+i*nmbrOfWlkrs];
    }
  }
}

__global__ void InitializeWalkersAtRandom ( const int nmbrOfWlkrs, const float dlt, Walker strtngWlkr, Walker *rndmWlkr, Walker *wlkrs, float *sttstcs )
{
  int wlIndx = threadIdx.x + blockDim.x * blockIdx.x;
  if ( wlIndx < nmbrOfWlkrs )
  {
    wlkrs[wlIndx] = AddWalkers ( strtngWlkr, ScaleWalker ( rndmWlkr[wlIndx], dlt ) );
    sttstcs[wlIndx] = 0;
  }
}

__global__ void InitializeWalkersAndStatisticsFromLastChain ( const int nmbrOfWlkrs, const float *lstChn, Walker *wlkrs, float *sttstcs, float *prrs )
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int p;
  if ( i < nmbrOfWlkrs )
  {
    p = 0;
    while ( p < NPRS )
    {
      wlkrs[i].par[p] = lstChn[p+i*(NPRS+2)];
      p += 1;
    }
    sttstcs[i] = lstChn[p+i*(NPRS+2)];
    p += 1;
    prrs[i] = lstChn[p+i*(NPRS+2)];
  }
}

__global__ void WriteWalkersAndStatisticsToChain ( const int nmbrOfWlkrs, const int stpIndx, const Walker *wlkrs, const float *sttstcs, const float *prrs, Walker *chnOfWlkrs, float *chnOfSttstcs, float *chnOfPrrs )
{
  int w = threadIdx.x + blockDim.x * blockIdx.x;
  int t = w + stpIndx * nmbrOfWlkrs;
  if ( w < nmbrOfWlkrs )
  {
    chnOfWlkrs[t] = wlkrs[w];
    chnOfSttstcs[t] = sttstcs[w];
    chnOfPrrs[t] = prrs[w];
  }
}

__global__ void AssembleArrayOfPriors ( const int nmbrOfWlkrs, const Walker *wlkrs, const float *nhMd, const float *nhSg, float *prrs )
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if ( i < nmbrOfWlkrs )
  {
    prrs[i] = PriorStatistic ( wlkrs[i], PriorCondition ( wlkrs[i] ), nhMd[i], nhSg[i] );
  }
}

__global__ void AssembleArrayOfAbsorptionFactors ( const int nmbrOfWlkrs, const int nmbrOfEnrgChnnls, const int nmbrOfElmnts, const float *crssctns, const float *abndncs, const int *atmcNmbrs, const Walker *wlkrs, float *absrptnFctrs )
{
  int enIndx = threadIdx.x + blockDim.x * blockIdx.x;
  int wlIndx = threadIdx.y + blockDim.y * blockIdx.y;
  int ttIndx = enIndx + wlIndx * nmbrOfEnrgChnnls;
  int elIndx, effElIndx, crIndx, prIndx;
  float xsctn, clmn, nh;
  if ( ( enIndx < nmbrOfEnrgChnnls ) && ( wlIndx < nmbrOfWlkrs ) )
  {
    if ( NHINDX == NPRS-1 )
    {
      elIndx = 0;
      prIndx = elIndx + NHINDX;
      crIndx = elIndx + enIndx * nmbrOfElmnts;
      effElIndx = atmcNmbrs[elIndx] - 1;
      nh = wlkrs[wlIndx].par[prIndx] * 1.E22;
      clmn = abndncs[effElIndx];
      xsctn = clmn * crssctns[crIndx];
      elIndx = 1;
      while ( elIndx < nmbrOfElmnts )
      {
        prIndx = elIndx + NHINDX;
        crIndx = elIndx + enIndx * nmbrOfElmnts;
        effElIndx = atmcNmbrs[elIndx] - 1;
        clmn = abndncs[effElIndx]; // * powf ( 10, wlkrs[wlIndx].par[prIndx] );
        xsctn = xsctn + clmn * crssctns[crIndx];
        elIndx += 1;
      }
      absrptnFctrs[ttIndx] = expf ( - nh * xsctn );
    }
    else if ( NHINDX == NPRS )
    {
      absrptnFctrs[ttIndx] = 1;
    }
  }
}

__global__ void AssembleArrayOfNoticedChannels ( const int nmbrOfChnnls, const float lwrNtcdEnrg, const float hghrNtcdEnrg, const float *lwrChnnlBndrs, const float *hghrChnnlBndrs, const float *gdQltChnnls, float *ntcdChnnls )
{
  int c = threadIdx.x + blockDim.x * blockIdx.x;
  if ( c < nmbrOfChnnls )
  {
    ntcdChnnls[c] = ( lwrChnnlBndrs[c] > lwrNtcdEnrg ) * ( hghrChnnlBndrs[c] < hghrNtcdEnrg ) * ( 1 - gdQltChnnls[c] );
  }
}

__global__ void AssembleArrayOfChannelStatistics ( const int nmbrOfWlkrs, const int nmbrOfChnnls, const float srcExptm, const float bckgrndExptm, const float backscal_src, const float backscal_bkg, const float *srcCnts, const float *bckgrndCnts, const float *flddMdlFlxs, float *chnnlSttstcs )
{
  int c = threadIdx.x + blockDim.x * blockIdx.x;
  int w = threadIdx.y + blockDim.y * blockIdx.y;
  int t = c + w * nmbrOfChnnls;
  if ( ( c < nmbrOfChnnls ) && ( w < nmbrOfWlkrs ) )
  {
    //chnnlSttstcs[t] = PoissonWithBackground ( srcCnts[c], bckgrndCnts[c], flddMdlFlxs[t], srcExptm, bckgrndExptm, backscal_src, backscal_bkg );
    chnnlSttstcs[t] = Poisson ( srcCnts[c], flddMdlFlxs[t], srcExptm );
  }
}

__global__ void AssembleArrayOfTimesStatistic ( const int nmbrOfWlkrs, const int nmbrOfPhtns, const float srcExptm, const Walker *wlk, const float *arrTms, float *tmsSttstcs )
{
  int a = threadIdx.x + blockDim.x * blockIdx.x;
  int w = threadIdx.y + blockDim.y * blockIdx.y;
  int t = a + w * nmbrOfPhtns;
  if ( ( a < nmbrOfPhtns ) && ( w < nmbrOfWlkrs ) )
  {
    tmsSttstcs[t] = GregoryLoredo ( arrTms[a], wlk[w], srcExptm, nmbrOfPhtns );
  }
}

__global__ void GenerateProposal ( const int nmbrOfHlfTheWlkrs, const int stpIndx, const int sbstIndx, const Walker *wlkrs, const float *rndmVls, float *zRndmVls, Walker *prpsdWlkrs, float *prpsdSttstcs )
{
  int wlIndx = threadIdx.x + blockDim.x * blockIdx.x;
  int ttSbIndx = wlIndx + sbstIndx * nmbrOfHlfTheWlkrs;
  int rnIndx, ttRnIndx, ttCmSbIndx, k;
  float zz;
  Walker B;
  if ( wlIndx < nmbrOfHlfTheWlkrs )
  {
    rnIndx = 0;
    ttRnIndx = wlIndx + rnIndx * nmbrOfHlfTheWlkrs + stpIndx * 3 * nmbrOfHlfTheWlkrs;
    zz = 0.5 * powf ( rndmVls[ttRnIndx] + 1, 2. );
    zRndmVls[wlIndx] = zz;
    rnIndx = 1;
    ttRnIndx = wlIndx + rnIndx * nmbrOfHlfTheWlkrs + stpIndx * 3 * nmbrOfHlfTheWlkrs;
    k = ( int ) truncf ( rndmVls[ttRnIndx] * ( nmbrOfHlfTheWlkrs - 1 + 0.999999 ) );
    ttCmSbIndx = k + ( 1 - sbstIndx ) * nmbrOfHlfTheWlkrs;
    B = AddWalkers ( wlkrs[ttSbIndx], ScaleWalker ( wlkrs[ttCmSbIndx], -1. ) );
    prpsdWlkrs[wlIndx] = AddWalkers ( wlkrs[ttCmSbIndx], ScaleWalker ( B, zz ) );
    prpsdSttstcs[wlIndx] = 0;
  }
}

__global__ void UpdateWalkers ( const int nmbrOfHlfTheWlkrs, const int stpIndx, const int sbstIndx, const Walker *prpsdWlkrs, const float *prpsdSttstcs, const float *prpsdPrrs, const float *zRndmVls, const float *rndmVls, Walker *wlkrs, float *sttstcs, float *prrs )
{
  int wlIndx = threadIdx.x + blockDim.x * blockIdx.x;
  int ttSbIndx = wlIndx + sbstIndx * nmbrOfHlfTheWlkrs;
  int rnIndx = 2;
  int ttRnIndx = wlIndx + rnIndx * nmbrOfHlfTheWlkrs + stpIndx * 3 * nmbrOfHlfTheWlkrs;
  float q;
  if ( wlIndx < nmbrOfHlfTheWlkrs )
  {
    q = - 0.5 * ( prpsdSttstcs[wlIndx] + prpsdPrrs[wlIndx] - sttstcs[ttSbIndx] - prrs[ttSbIndx] );
    q = expf ( q ) * powf ( zRndmVls[wlIndx], NPRS - 1 );
    if ( q > rndmVls[ttRnIndx] )
    {
      wlkrs[ttSbIndx] = prpsdWlkrs[wlIndx];
      sttstcs[ttSbIndx] = prpsdSttstcs[wlIndx];
      prrs[ttSbIndx] = prpsdPrrs[wlIndx];
     }
  }
}

__global__ void ComplexPointwiseMultiplyByConjugateAndScale ( const int nmbrOfStps, const int nmbrOfWlkrs, const float scl, Complex *a )
{
  int s = threadIdx.x + blockDim.x * blockIdx.x;
  int w = threadIdx.y + blockDim.y * blockIdx.y;
  int t = s + w * nmbrOfStps;
  if ( ( w < nmbrOfWlkrs ) && ( s < nmbrOfStps ) )
  {
    a[t] = ScaleComplex ( MultiplyComplex ( a[t], ConjugateComplex ( a[t] ) ), scl );
  }
}

__global__ void ReturnChainFunctionTest ( const int nmbrOfStps, const int nmbrOfWlkrs, const int sw, float *chn, Complex *a  )
{
  int wlIndx = threadIdx.x + blockDim.x * blockIdx.x;
  int stIndx = threadIdx.y + blockDim.y * blockIdx.y;
  int ttIndx0 = wlIndx + stIndx * nmbrOfWlkrs;
  int ttIndx1 = stIndx + wlIndx * nmbrOfStps;
  if ( ( wlIndx < nmbrOfWlkrs ) && ( stIndx < nmbrOfStps ) )
  {
    if ( sw == 0 ) { a[ttIndx1].x = chn[ttIndx0]; a[ttIndx1].y = 0; }
    else if ( sw == 1 ) { chn[ttIndx0] = a[ttIndx1].x; }
  }
}

__global__ void ReturnChainFunction ( const int nmbrOfStps, const int nmbrOfWlkrs, const int prmtrIndx, const Walker *chnOfWlkrs, float *chnFnctn )
{
  int w = threadIdx.x + blockDim.x * blockIdx.x;
  int s = threadIdx.y + blockDim.y * blockIdx.y;
  int t = w + s * nmbrOfWlkrs;
  if ( ( w < nmbrOfWlkrs ) && ( s < nmbrOfStps ) )
  {
    chnFnctn[t] = chnOfWlkrs[t].par[prmtrIndx];
  }
}

__global__ void ReturnConstantArray ( const int N, const float c, float *a )
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if ( i < N ) { a[i] = c; }
}

__global__ void ReturnCentralChainFunction ( const int nmbrOfStps, const int nmbrOfWlkrs, const float *smOfChnFnctn, const float *chnFnctn, float *cntrlChnFnctn )
{
  int w = threadIdx.x + blockDim.x * blockIdx.x;
  int s = threadIdx.y + blockDim.y * blockIdx.y;
  int t = w + s * nmbrOfWlkrs;
  if ( ( w < nmbrOfWlkrs ) && ( s < nmbrOfStps )  )
  {
    cntrlChnFnctn[t] = chnFnctn[t] - smOfChnFnctn[w];
  }
}

__global__ void NormalizeChain ( const int nmbrOfStps, float *chn )
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if ( i < nmbrOfStps ) { chn[i] = chn[i] / chn[0]; }
}

__global__ void MakeMatrix ( const int nmbrOfStps, const float *chn, float *cmSmMtrx )
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int j = threadIdx.y + blockDim.y * blockIdx.x;
  if ( ( i < nmbrOfStps ) && ( j < nmbrOfStps ) )
  {
    cmSmMtrx[i+j*nmbrOfStps] = ( i <= j ) * chn[i];
  }
}

__global__ void BilinearInterpolation ( const int nmbrOfWlkrs, const int nmbrOfEnrgChnnls, const int tIndx, const int grIndx, const float *data, const float *xin, const float *yin, const int M1, const int M2, const float *enrgChnnls, const Walker *wlkrs, float *mdlFlxs )
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int j = threadIdx.y + blockDim.y * blockIdx.y;
  float xxout, yyout, sa, gr, a, b, d00, d01, d10, d11, tmp1, tmp2, tmp3;
  int v, w;
  if ( ( i < nmbrOfEnrgChnnls ) && ( j < nmbrOfWlkrs ) )
  {
    gr = sqrtf ( 1. - 2.952 * MNS / RNS );
    sa = powf ( RNS, 2. );
    xxout = log10f ( enrgChnnls[i] / gr );
    yyout = wlkrs[j].par[tIndx];
    v = FindElementIndex ( xin, M1, xxout );
    w = FindElementIndex ( yin, M2, yyout );
    a = ( xxout - xin[v] ) / ( xin[v+1] - xin[v] );
    b = ( yyout - yin[w] ) / ( yin[w+1] - yin[w] );
    if ( v < M1 && w < M2 ) d00 = data[w*M1+v]; else d00 = 0.;
    if ( v+1 < M1 && w < M2 ) d10 = data[w*M1+v+1]; else d10 = 0;
    if ( v < M1 && w+1 < M2 ) d01 = data[(w+1)*M1+v]; else d01 = 0;
    if ( v+1 < M1 && w+1 < M2 ) d11 = data[(w+1)*M1+v+1]; else d11 = 0;
    tmp1 = a * d10 + ( -d00 * a + d00 );
    tmp2 = a * d11 + ( -d01 * a + d01 );
    tmp3 = b * tmp2 + ( -tmp1 * b + tmp1 );
    mdlFlxs[i+j*nmbrOfEnrgChnnls] = powf ( 10., tmp3 ) * sa;
  }
}

__global__ void BilinearInterpolationNsmax ( const int nmbrOfWlkrs, const int nmbrOfEnrgChnnls, const int tIndx, const int grIndx, const float *data, const float *xin, const float *yin, const int M1, const int M2, const float *enrgChnnls, const Walker *wlkrs, float *mdlFlxs )
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int j = threadIdx.y + blockDim.y * blockIdx.y;
  float xxout, yyout, sa, gr, a, b, d00, d01, d10, d11, tmp1, tmp2, tmp3;
  int v, w;
  if ( ( i < nmbrOfEnrgChnnls ) && ( j < nmbrOfWlkrs ) )
  {
    gr = sqrtf ( 1. - 2.952 * MNS / RNS );
    sa = powf ( RNS, 2. );
    xxout = log10f ( enrgChnnls[i] / gr );
    yyout = wlkrs[j].par[tIndx];
    v = FindElementIndex ( xin, M1, xxout );
    w = FindElementIndex ( yin, M2, yyout );
    a = ( xxout - xin[v] ) / ( xin[v+1] - xin[v] );
    b = ( yyout - yin[w] ) / ( yin[w+1] - yin[w] );
    if ( v < M1 && w < M2 ) d00 = data[w*M1+v]; else d00 = 0.;
    if ( v+1 < M1 && w < M2 ) d10 = data[w*M1+v+1]; else d10 = 0;
    if ( v < M1 && w+1 < M2 ) d01 = data[(w+1)*M1+v]; else d01 = 0;
    if ( v+1 < M1 && w+1 < M2 ) d11 = data[(w+1)*M1+v+1]; else d11 = 0;
    tmp1 = a * d10 + ( -d00 * a + d00 );
    tmp2 = a * d11 + ( -d01 * a + d01 );
    tmp3 = b * tmp2 + ( -tmp1 * b + tmp1 );
    mdlFlxs[i+j*nmbrOfEnrgChnnls] = powf ( 10., tmp3 + 26.1787440 - xxout ) * sa;
  }
}

__global__ void LinearInterpolation ( const int nmbrOfWlkrs, const int nmbrOfDistBins, const int dIndx, const float *Dist, const float *EBV, const float *errEBV, const Walker *wlkrs, float *mNh, float *sNh )
{
  int w = threadIdx.x + blockDim.x * blockIdx.x;
  float xxout, a, dmNh0, dmNh1, dsNh0, dsNh1, tmpMNh, tmpSNh;
  int v;
  if ( w < nmbrOfWlkrs )
  {
    xxout = wlkrs[w].par[dIndx];
    v = FindElementIndex ( Dist, nmbrOfDistBins, xxout );
    a = ( xxout - Dist[v] ) / ( Dist[v+1] - Dist[v] );
    if ( v < nmbrOfDistBins ) dmNh0 = EBV[v]; else dmNh0 = 0;
    if ( v+1 < nmbrOfDistBins ) dmNh1 = EBV[v+1]; else dmNh1 = 0;
    tmpMNh = a * dmNh1 + ( -dmNh0 * a + dmNh0 );
    if ( v < nmbrOfDistBins ) dsNh0 = errEBV[v]; else dsNh0 = 0;
    if ( v+1 < nmbrOfDistBins ) dsNh1 = errEBV[v+1]; else dsNh1 = 0;
    tmpSNh = a * dsNh1 + ( -dsNh0 * a + dsNh0 );
    tmpMNh = powf ( 10, tmpMNh );
    tmpSNh = powf ( 10, tmpSNh );
    mNh[w] = 0.8 * tmpMNh;
    sNh[w] = 0.8 * tmpMNh * ( powf ( tmpSNh / tmpMNh, 2 ) + powf ( 0.3 / 0.8, 2 ) );
  }
}

__global__ void LinearInterpolationNoErrors ( const int nmbrOfWlkrs, const int nmbrOfDistBins, const int dIndx, const float *Dist, const float *EBV, const Walker *wlkrs, float *mNh, float *sNh )
{
  int w = threadIdx.x + blockDim.x * blockIdx.x;
  float xxout, a, dmNh0, dmNh1, tmpMNh;
  int v;
  if ( w < nmbrOfWlkrs )
  {
    xxout = wlkrs[w].par[dIndx];
    v = FindElementIndex ( Dist, nmbrOfDistBins, xxout );
    a = ( xxout - Dist[v] ) / ( Dist[v+1] - Dist[v] );
    if ( v < nmbrOfDistBins ) dmNh0 = EBV[v]; else dmNh0 = 0;
    if ( v+1 < nmbrOfDistBins ) dmNh1 = EBV[v+1]; else dmNh1 = 0;
    tmpMNh = a * dmNh1 + ( -dmNh0 * a + dmNh0 );
    tmpMNh = powf ( 10, tmpMNh );
    mNh[w] = 0.7 * tmpMNh;
    sNh[w] = 0.7 * tmpMNh * 0.1;
  }
}

__host__ int ReadFitsInfo ( const char *spcFl, int *nmbrOfEnrgChnnls, int *nmbrOfChnnls, int *nmbrOfRmfVls, float *srcExptm, float *bckgrndExptm, char srcTbl[FLEN_CARD], char arfTbl[FLEN_CARD], char rmfTbl[FLEN_CARD], char bckgrndTbl[FLEN_CARD] )
{
  fitsfile *ftsPntr;       /* pointer to the FITS file; defined in fitsio.h */
  int status = 0, intnull = 0, anynull = 0, colnum;
  char card[FLEN_CARD], colNgr[] = "N_GRP", colNch[] = "N_CHAN";
  float floatnull;
  /* Open Spectrum  */
  snprintf ( srcTbl, sizeof ( card ), "%s%s", spcFl, "[SPECTRUM]" );
  fits_open_file ( &ftsPntr, srcTbl, READONLY, &status );
  fits_read_key ( ftsPntr, TINT, "NAXIS2", nmbrOfChnnls, NULL, &status );
  fits_read_key ( ftsPntr, TFLOAT, "EXPOSURE", srcExptm, NULL, &status );
  /* Read names of arf rmf and background */
  fits_read_key ( ftsPntr, TSTRING, "ANCRFILE", card, NULL, &status );
  snprintf ( arfTbl, sizeof ( card ), "%s%s", card, "[SPECRESP]" );
  fits_read_key ( ftsPntr, TSTRING, "RESPFILE", card, NULL, &status );
  snprintf ( rmfTbl, sizeof ( card ), "%s%s", card, "[MATRIX]" );
  /* Open Background file */
  fits_read_key ( ftsPntr, TSTRING, "BACKFILE", card, NULL, &status );
  snprintf ( bckgrndTbl, sizeof ( card ), "%s%s", card, "[SPECTRUM]" );
  fits_open_file ( &ftsPntr, bckgrndTbl, READONLY, &status );
  if ( status == 0 && BACKIN == 1 )
  {
    fits_read_key ( ftsPntr, TFLOAT, "EXPOSURE", bckgrndExptm, NULL, &status );
  }
  else
  {
    *bckgrndExptm = 0.0;
    status = 0;
  }
  /* Open RMF file */
  fits_open_file ( &ftsPntr, rmfTbl, READONLY, &status );
  if ( status != 0 ) { printf ( " Error: Opening rmf table fails\n" ); return 1; }
  fits_read_key ( ftsPntr, TINT, "NAXIS2", nmbrOfEnrgChnnls, NULL, &status );
  if ( status != 0 ) { printf ( " Error: Reading NAXIS2 key from rmf table fails\n" ); return 1; }
  int *n_grp;
  n_grp = ( int * ) malloc ( *nmbrOfEnrgChnnls * sizeof ( int ) );
  fits_get_colnum ( ftsPntr, CASEINSEN, colNgr, &colnum, &status );
  fits_read_col_int ( ftsPntr, colnum, 1, 1, *nmbrOfEnrgChnnls, intnull, n_grp, &anynull, &status );
  int *n_chan_vec;
  n_chan_vec = ( int * ) malloc ( *nmbrOfChnnls * sizeof ( int ) );
  int sum = 0;
  for ( int i = 0; i < *nmbrOfEnrgChnnls; i++ )
  {
    fits_get_colnum ( ftsPntr, CASEINSEN, colNch, &colnum, &status );
    fits_read_col ( ftsPntr, TINT, colnum, i+1, 1, n_grp[i], &floatnull, n_chan_vec, &anynull, &status );
    for ( int j = 0; j < n_grp[i]; j++ )
    {
      sum = sum + n_chan_vec[j];
    }
  }
  *nmbrOfRmfVls = sum;
  free ( n_chan_vec );
  free ( n_grp );
  return 0;
}

__host__ int ReadFitsData ( const int verbose, const char srcTbl[FLEN_CARD], const char arfTbl[FLEN_CARD], const char rmfTbl[FLEN_CARD], const char bckgrndTbl[FLEN_CARD], const int nmbrOfEnrgChnnls, const int nmbrOfChnnls, const int nmbrOfRmfVls, float *backscal_src, float *backscal_bkg, float *srcCnts, float *bckgrndCnts, float *arfFctrs, float *rmfVlsInCsc, int *rmfIndxInCsc, int *rmfPntrInCsc, float *gdQltChnnls, float *lwrChnnlBndrs, float *hghrChnnlBndrs, float *enrgChnnls )
{
  fitsfile *ftsPntr;       /* pointer to the FITS file; defined in fitsio.h */
  int status = 0, anynull, colnum, intnull = 0, rep_chan = 100;
  char card[FLEN_CARD], EboundsTable[FLEN_CARD], Telescop[FLEN_CARD];
  char colNgr[]="N_GRP", colNch[]="N_CHAN",  colFch[]="F_CHAN", colCounts[]="COUNTS", colSpecResp[]="SPECRESP", colEnLo[]="ENERG_LO", colEnHi[]="ENERG_HI", colMat[]="MATRIX", colEmin[]="E_MIN", colEmax[]="E_MAX";
  float floatnull;
  /* Read Spectrum: */
  fits_open_file ( &ftsPntr, srcTbl, READONLY, &status );
  fits_read_key ( ftsPntr, TSTRING, "RESPFILE", card, NULL, &status );
  snprintf ( EboundsTable, sizeof ( EboundsTable ), "%s%s", card, "[EBOUNDS]" );
  fits_read_key ( ftsPntr, TFLOAT, "BACKSCAL", backscal_src, NULL, &status );
  fits_read_key ( ftsPntr, TSTRING, "TELESCOP", Telescop, NULL, &status );
  fits_get_colnum ( ftsPntr, CASEINSEN, colCounts, &colnum, &status );
  fits_read_col ( ftsPntr, TFLOAT, colnum, 1, 1, nmbrOfChnnls, &floatnull, srcCnts, &anynull, &status );
  /* Read ARF FILE: */
  fits_open_file ( &ftsPntr, arfTbl, READONLY, &status );
  fits_get_colnum ( ftsPntr, CASEINSEN, colSpecResp, &colnum, &status );
  fits_read_col ( ftsPntr, TFLOAT, colnum, 1, 1, nmbrOfEnrgChnnls, &floatnull, arfFctrs, &anynull, &status );
  /* Read Background: */
  fits_open_file ( &ftsPntr, bckgrndTbl, READONLY, &status );
  if ( status == 0 && BACKIN == 1 )
  {
    fits_read_key ( ftsPntr, TFLOAT, "BACKSCAL", backscal_bkg, NULL, &status );
    fits_get_colnum ( ftsPntr, CASEINSEN, colCounts, &colnum, &status );
    fits_read_col ( ftsPntr, TFLOAT, colnum, 1, 1, nmbrOfChnnls, &floatnull, bckgrndCnts, &anynull, &status );
  }
  else
  {
    if ( verbose == 1)
    {
      printf ( " Warning: Background table is not used, background exposure and background are set to 0.\n " );
    }
    for ( int i = 0; i < nmbrOfChnnls; i++ )
    {
      bckgrndCnts[i] = 0;
    }
    status = 0;
  }
  /* Read RMF file */
  fits_open_file ( &ftsPntr, rmfTbl, READONLY, &status );
  float *enelo_vec, *enehi_vec;
  enelo_vec = ( float * ) malloc ( nmbrOfEnrgChnnls * sizeof ( float ) );
  enehi_vec = ( float * ) malloc ( nmbrOfEnrgChnnls * sizeof ( float ) );
  fits_get_colnum ( ftsPntr, CASEINSEN, colEnLo, &colnum, &status );
  fits_read_col ( ftsPntr, TFLOAT, colnum, 1, 1, nmbrOfEnrgChnnls, &floatnull, enelo_vec, &anynull, &status );
  fits_get_colnum ( ftsPntr, CASEINSEN, colEnHi, &colnum, &status );
  fits_read_col ( ftsPntr, TFLOAT, colnum, 1, 1, nmbrOfEnrgChnnls, &floatnull, enehi_vec, &anynull, &status );
  for ( int i = 0; i < nmbrOfEnrgChnnls; i++ )
  {
    enrgChnnls[i] = enelo_vec[i];
  }
  enrgChnnls[nmbrOfEnrgChnnls] = enehi_vec[nmbrOfEnrgChnnls-1];
  int *f_chan_vec, *n_chan_vec;
  f_chan_vec = ( int * ) malloc ( rep_chan * sizeof ( int ) );
  n_chan_vec = ( int * ) malloc ( rep_chan * sizeof ( int ) );
  int *f_chan, *n_chan;
  f_chan = ( int * ) malloc ( rep_chan * nmbrOfEnrgChnnls * sizeof ( int ) );
  n_chan = ( int * ) malloc ( rep_chan * nmbrOfEnrgChnnls * sizeof ( int ) );
  int *n_grp;
  n_grp = ( int * ) malloc ( nmbrOfEnrgChnnls * sizeof ( int ) );
  fits_get_colnum ( ftsPntr, CASEINSEN, colNgr, &colnum, &status );
  fits_read_col_int ( ftsPntr, colnum, 1, 1, nmbrOfEnrgChnnls, intnull, n_grp, &anynull, &status );
  for ( int i = 0; i < nmbrOfEnrgChnnls; i++ )
  {
    fits_get_colnum ( ftsPntr, CASEINSEN, colNch, &colnum, &status );
    fits_read_col_int ( ftsPntr, colnum, i+1, 1, n_grp[i], intnull, n_chan_vec, &anynull, &status );
    for ( int j = 0; j < rep_chan; j++ )
    {
      n_chan[i*rep_chan+j] = n_chan_vec[j];
    }
  }
  for ( int i = 0; i < nmbrOfEnrgChnnls; i++ )
  {
    fits_get_colnum ( ftsPntr, CASEINSEN, colFch, &colnum, &status );
    fits_read_col ( ftsPntr, TINT, colnum, i+1, 1, n_grp[i], &floatnull, f_chan_vec, &anynull, &status );
    for ( int j = 0; j < rep_chan; j++ )
    {
      f_chan[i*rep_chan+j] = f_chan_vec[j];
    }
  }
  int sum = 0;
  rmfPntrInCsc[0] = 0;
  for ( int i = 0; i < nmbrOfEnrgChnnls; i++ )
  {
    for ( int j = 0; j < n_grp[i]; j++ )
    {
      sum = sum + n_chan[rep_chan*i+j];
    }
    rmfPntrInCsc[i+1] = sum;
  }
  int m = 0;
  if ( nmbrOfChnnls != 1024 )
  {
    for ( int i = 0; i < nmbrOfEnrgChnnls; i++ )
    {
      for ( int j = 0; j < n_grp[i]; j++ )
      {
        for ( int k = f_chan[rep_chan*i+j] ; k < f_chan[rep_chan*i+j] + n_chan[rep_chan*i+j]; k++ )
        {
          rmfIndxInCsc[m] = k;
          m = m + 1;
        }
      }
    }
  }
  else if ( nmbrOfChnnls == 1024 )
  {
    for ( int i = 0; i < nmbrOfEnrgChnnls; i++ )
    {
      for ( int j = 0; j < n_grp[i]; j++ )
      {
        for ( int k = f_chan[rep_chan*i+j] - 1; k < f_chan[rep_chan*i+j] - 1 + n_chan[rep_chan*i+j]; k++ )
        {
          rmfIndxInCsc[m] = k;
          m = m + 1;
        }
      }
    }
  }
  float *rmf_vec;
  rmf_vec = ( float * ) malloc ( nmbrOfChnnls * sizeof ( float ) );
  fits_get_colnum ( ftsPntr, CASEINSEN, colMat, &colnum, &status );
  m = 0;
  for ( int i = 0; i < nmbrOfEnrgChnnls; i++ )
  {
    sum = rmfPntrInCsc[i+1] - rmfPntrInCsc[i];
    fits_read_col ( ftsPntr, TFLOAT, colnum, i+1, 1, sum, &floatnull, rmf_vec, &anynull, &status );
    for ( int k = 0; k < sum; k++ )
    {
      rmfVlsInCsc[m] = rmf_vec[k];
      m = m + 1;
    }
  }
  /* Read Ebounds Table: */
  fits_open_file ( &ftsPntr, EboundsTable, READONLY, &status );
  fits_get_colnum ( ftsPntr, CASEINSEN, colEmin, &colnum, &status );
  fits_read_col ( ftsPntr, TFLOAT, colnum, 1, 1, nmbrOfChnnls, &floatnull, lwrChnnlBndrs, &anynull, &status );
  fits_get_colnum ( ftsPntr, CASEINSEN, colEmax, &colnum, &status );
  fits_read_col ( ftsPntr, TFLOAT, colnum, 1, 1, nmbrOfChnnls, &floatnull, hghrChnnlBndrs, &anynull, &status );
  free ( enelo_vec );
  free ( enehi_vec );
  free ( rmf_vec );
  free ( f_chan_vec );
  free ( n_chan_vec );
  free ( n_chan );
  free ( f_chan );
  free ( n_grp );
  return 0;
}

#endif // _STRCTRSANDFNCTNS_CU_
