#ifndef _GWMCMCCUDA_CU_
#define _GWMCMCCUDA_CU_

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <fitsio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include "StrctrsAndFnctns.cuh"

int main ( int argc, char *argv[] ) {
  const int vrb = 1;

  Cupar cdp[1];
  cdp[0].dev = atoi ( argv[1] );

  initializeCuda ( cdp );

  if ( vrb ) {
    printf ( "\n" );
    printf ( ".................................................................\n" );
    printf ( " CUDA device ID: %d\n", cdp[0].dev );
    printf ( " CUDA device Name: %s\n", cdp[0].prop.name );
    printf ( " Driver API: v%d \n", cdp[0].driverVersion[0] );
    printf ( " Runtime API: v%d \n", cdp[0].runtimeVersion[0] );
  }

  Model mdl[1];
  InitializeModel ( mdl );

  Chain chn[1];
  Spectrum spc[NSPCTR];
  Spectrum bkg[NSPCTR];
  for ( int i = 0; i < NSPCTR; i++ ) {
    spc[i].name = argv[2+2*i];
    bkg[i].name = argv[2+2*i+1];
  }

  chn[0].name = argv[2*NSPCTR+2];
  chn[0].nwl = atoi ( argv[2*NSPCTR+3] );
  chn[0].nst = atoi ( argv[2*NSPCTR+4] );
  chn[0].indx = atoi ( argv[2*NSPCTR+5] );
  chn[0].dim = NPRS;
  chn[0].dim1 = chn[0].dim + 3;
  chn[0].dlt = 1.E-4;
  chn[0].nkb = 100;

  const float lwrNtcdEnrg1 = ( float ) atof ( argv[2*NSPCTR+6] );
  const float hghrNtcdEnrg1 = ( float ) atof ( argv[2*NSPCTR+7] );

  for ( int i = 0; i < NSPCTR; i++ ) {
    spc[i].lwrNtcdEnrg = lwrNtcdEnrg1;
    spc[i].hghrNtcdEnrg = hghrNtcdEnrg1;
  }

  specData ( cdp, vrb, mdl, spc, bkg );

  allocateChain ( chn );

  chn[0].x0[0] = 5.9;
  chn[0].xbnd[0] = 5.5;
  chn[0].xbnd[1] = 6.5;

  chn[0].x0[1] = 0.0;
  chn[0].xbnd[2] = -25.;
  chn[0].xbnd[3] = 25.;

  chn[0].x0[2] = 1.5;
  chn[0].xbnd[4] = -25.;
  chn[0].xbnd[5] = 25.;

  chn[0].x0[3] = -5.;
  chn[0].xbnd[6] = -25.;
  chn[0].xbnd[7] = 25.;

  chn[0].x0[4] = 1.5;
  chn[0].xbnd[8] = -25.;
  chn[0].xbnd[9] = 2.;

  chn[0].x0[5] = -5.;
  chn[0].xbnd[10] = -25.;
  chn[0].xbnd[11] = 25.;

  chn[0].x0[6] = 1.5;
  chn[0].xbnd[12] = -25.;
  chn[0].xbnd[13] = 25.;

  chn[0].x0[7] = -5.;
  chn[0].xbnd[14] = -25.;
  chn[0].xbnd[15] = 25.;

  chn[0].x0[8] = 0.1;
  chn[0].xbnd[16] = 0.;
  chn[0].xbnd[17] = 25.;

  initializeChain ( cdp, chn );
  modelStatistic0 ( cdp, mdl, chn, spc, bkg );

  if ( vrb ) {
    printf ( ".................................................................\n" );
    printf ( " Start ...                                                  \n" );
  }

  hipEventRecord ( cdp[0].start, 0 );

  initializeRandomForStreach ( cdp, chn );
  chn[0].ist = 0;
  while ( chn[0].ist < chn[0].nst ) {
    chn[0].isb = 0;
    while ( chn[0].isb < 2 ) {
      streachMove ( cdp, chn );
      modelStatistic1 ( cdp, mdl, chn, spc, bkg );
      streachUpdate ( cdp, chn, mdl );
      chn[0].isb += 1;
    }
    saveCurrent ( chn );
    chn[0].ist += 1;
  }

  chainMomentsAndKde ( cdp, chn );

  if ( vrb ) {
    printf ( " Done!\n" );
  }

  hipEventRecord ( cdp[0].stop, 0 );
  hipEventSynchronize ( cdp[0].stop );
  hipEventElapsedTime ( &chn[0].time, cdp[0].start, cdp[0].stop );

  if ( vrb ) {
    printf ( ".................................................................\n" );
    printf ( " Time to generate -- %3.1f ms\n", chn[0].time );
  }

  hipEventRecord ( cdp[0].start, 0 );

  averagedAutocorrelationFunction ( cdp, chn );

  hipEventRecord ( cdp[0].stop, 0 );
  hipEventSynchronize ( cdp[0].stop );
  hipEventElapsedTime ( &chn[0].time, cdp[0].start, cdp[0].stop );

  if ( vrb ) {
    printf ( ".................................................................\n" );
    printf ( " Autocorrelation time window             -- %i\n", chn[0].mmm );
    printf ( " Autocorrelation time                    -- %.8E\n", chn[0].atcTime );
    printf ( " Autocorrelation time threshold          -- %.8E\n", chn[0].nst / 5e1f );
    printf ( " Effective number of independent samples -- %.8E\n", chn[0].nwl * chn[0].nst / chn[0].atcTime );
    printf ( " Time to compute acor time               -- %3.1f ms\n", chn[0].time );
  }

  sortQKde ( chn );

  if ( vrb ) {
    printf ( ".................................................................\n" );
    printf ( " Medium                    -- " );
    for ( int i = 0; i < chn[0].dim1; i++ ) {
      printf ( " %2.2f ", chn[0].msmp[i] );
    }
    printf ( "\n" );
    printf ( " Std. deviation            -- " );
    for ( int i = 0; i < chn[0].dim1; i++ ) {
      printf ( " %2.2f ", chn[0].stdsmp[i] );
    }
    printf ( "\n" );
    printf ( " Max pdf (best-fit) values -- " );
    for ( int j = 0; j < chn[0].dim1; j++ ) {
        printf ( " %2.2f ", chn[0].skbin[j+0*chn[0].dim1] );
    }
    printf ( "\n" );
  }

  for ( int i = 0; i < chn[0].dim; i++ ) {
    chn[0].xx[i] = chn[0].skbin[i];
    printf ( " %2.2f ", chn[0].xx[i] );
  }
  printf ( "\n" );

  chn[0].didi[0] = chn[0].skbin[chn[0].dim];

  modelStatistic0 ( cdp, mdl, chn, spc, bkg );

  hipDeviceSynchronize ();

  /* Write results to a file */
  simpleWriteDataFloat ( "Autocor.out", chn[0].nst, chn[0].atcrrFnctn );
  simpleWriteDataFloat ( "AutocorCM.out", chn[0].nst, chn[0].cmSmAtCrrFnctn );
  writeSpectraToFile ( "Spectra.out", spc );
  writeWhalesToFile ( chn[0].name, chn[0].indx, chn[0].dim1, chn[0].nwl*chn[0].nst, chn[0].whales );
  writeChainToFile ( chn[0].name, chn[0].indx, chn[0].dim, chn[0].nwl, chn[0].nst, chn[0].smpls, chn[0].stat, chn[0].priors, chn[0].dist, chn[0].chiTwo );

  destroyCuda ( cdp );
  freeChain ( chn );
  FreeModel ( mdl );
  FreeSpec ( spc );

  // Reset the device and exit
  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application exits

  cdp[0].err = hipDeviceReset ();
  if ( cdp[0].err != hipSuccess ) {
    fprintf ( stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString ( cdp[0].err ) );
    exit ( EXIT_FAILURE );
  }

  return 0;
}

#endif // _GWMCMCCUDA_CU_
