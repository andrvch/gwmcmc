#include "hip/hip_runtime.h"
#ifndef _GWMCMCCUDA_CU_
#define _GWMCMCCUDA_CU_

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <fitsio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include "StrctrsAndFnctns.cuh"

__global__ void shiftWalkers ( const int dim, const int nwl, const float *xx, const float *x, float *yy ) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int j = threadIdx.y + blockDim.y * blockIdx.y;
  int t = i + j * dim;
  if ( i < dim && j < nwl ) {
    yy[t] = xx[t] - x[i];
  }
}

__global__ void addWalkers ( const int dim, const int nwl, const float *xx0, const float *xxW, float *xx1 ) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int j = threadIdx.y + blockDim.y * blockIdx.y;
  int t = i + j * dim;
  if ( i < dim && j < nwl ) {
    xx1[t] = xx0[t] + xxW[t];
  }
}

__global__ void sliceArray ( const int n, const int indx, const float *ss, float *zz ) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if ( i < n ) {
    zz[i] = ss[i+indx];
  }
}

__host__ void proposeWalkMove ( const Cuparam *cdp, Chain *chn ) {
  int incxx = INCXX, incyy = INCYY;
  float alpha = ALPHA, beta = BETA;
  int n, indx;
  n = chn[0].dim * chn[0].nwl / 2
  dim3 bl ( THRDSPERBLCK );
  dim3 gr ( ( n + bl.x - 1 ) / bl.x );
  indx = chn[0].isb * chn[0].nwl / 2 * chn[0].dim;
  sliceArray <<< gr, bl >>> ( n, indx, chn[0].xx, chn[0].xx0 );
  indx = ( 1 - chn[0].isb ) * chn[0].nwl / 2 * chn[0].dim;
  sliceArray <<< gr, bl >>> ( n, indx, chn[0].xx, chn[0].xxC );
  n = chn[0].nwl / 2 * chn[0].nwl / 2;
  indx = chn[0].ist * 2 * n + chn[0].isb * n;
  gr.x = ( n + bl.x - 1 ) / bl.x;
  sliceArray <<< gr, bl >>> ( n, indx, chn[0].stn, chn[0].zz );
  hipblasSgemv ( cdp[0].cublasHandle, HIPBLAS_OP_N, chn[0].dim, chn[0].nwl/2, &alpha, chn[0].xxC, chn[0].dim, chn[0].x1, incxx, &beta, chn[0].xCM, incyy );
  dim3 bl1 ( THRDSPERBLCK, THRDSPERBLCK );
  dim3 gr1 ( ( chn[0].dim + bl1.x - 1 ) / bl1.x, ( chn[0].nwl/2 + bl1.y - 1 ) / bl1.y );
  shiftWalkers <<< gr1, bl1 >>> ( chn[0].dim, chn[0].nwl/2, chn[0].xxC, chn[0].xCM, chn[0].xxCM );
  hipblasSgemm ( cdp[0].cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, chn[0].dim, chn[0].nwl/2 , chn[0].nwl/2, &alpha, chn[0].xxCM, chn[0].dim, chn[0].zz, chn[0].nwl/2, &beta, chn[0].xxW, chn[0].dim );
  addWalkers <<< gr1, bl1 >>> ( chn[0].dim, chn[0].nwl/2, chn[0].xx0, chn[0].xxW, chn[0].xx1 );
}

__host__ __device__ int PriorCondition ( const Walker w ) {
  int cnd = 1;
  //for ( int i = 0; i <  NPRS; i++ ) {
  //  cnd *=  0. < w.par[i];
  //}
  return cnd;
}

__host__ __device__ float PriorStatistic ( const Walker w, const int cnd ) {
  float p = 0, sum = 0;
  if ( cnd ) {
    p = sum;
  } else {
    p = INF;
  }
  return p;
}

__global__ void AssembleArrayOfStatistic ( const int dim, const int n, const float *xx, float *s ) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if ( i < n ) {
    s[i] = pow ( xx[i*dim] - xx[1+i*dim], 2. ) / 0.1 + pow ( xx[i*dim] + xx[1+i*dim], 2. );
  }
}

__host__ int Statistics ( const int n, const float *xx, float *s ) {
  AssembleArrayOfStatistic <<< Blocks ( n ), THRDSPERBLCK >>> ( n, xx, s );
  return 0;
}

__global__ void AssembleArrayOfPriors ( const int n, const Walker *w, float *p ) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if ( i < n ) {
    p[i] = PriorStatistic ( w[i], PriorCondition ( w[i] ) );
  }
}

__host__ int Priors ( const int n, const Walker *wlk, float *prr ) {
  AssembleArrayOfPriors <<< Blocks ( n ), THRDSPERBLCK >>> ( n, wlk, prr );
  return 0;
}

/**
 * Host main routine
 */
int main ( int argc, char *argv[] ) {
  dim3 dimBlock ( THRDSPERBLCK, THRDSPERBLCK );
  const int verbose = 1;
  const float dlt = 1.E-6;
  const float p0[NPRS] = { 0.7, 1.2 };

  Cuparam cdp[1];
  Chain chn[1];

  cdp[0].dev = atoi( argv[1] );
  chn[0].thrdNm = argv[2];
  chn[0].nmbrOfWlkrs = atoi ( argv[3] );
  chn[0].nWlk = chn[0].nmbrOfWlkrs;
  chn[0].nmbrOfStps = atoi ( argv[4] );
  chn[0].thrdIndx = atoi ( argv[5] );
  chn[0].dlt = dlt;
  chn[0].dimWlk = 2;

  InitializeCuda ( verbose, cdp );
  InitializeChain ( verbose, cdp, p0, chn );

  hiprandGenerateUniform ( cdp[0].curandGnrtr, chn[0].rndmVls, NPRS * chn[0].nmbrOfWlkrs );

  if ( chn[0].thrdIndx == 0 ) {
    InitAtRandom ( chn );
    Priors ( chn[0].nmbrOfWlkrs, chn[0].wlkrs, chn[0].prrs );
    Statistics ( chn[0].nmbrOfWlkrs, chn[0].wlkrs, chn[0].sttstcs );
  } else {
    InitFromLast ( chn );
  }

  hipEventRecord ( cdp[0].start, 0 );

  printf ( ".................................................................\n" );
  printf ( " Start ...                                                  \n" );

  hiprandGenerateUniform ( cdp[0].curandGnrtr, chn[0].rndmVls, chn[0].nmbrOfStps * 2 * chn[0].nmbrOfWlkrs / 2 );
  hiprandGenerateNormal ( cdp[0].curandGnrtr, chn[0].stnrm, chn[0].nmbrOfStps * 2 * chn[0].nmbrOfWlkrs / 2 * chn[0].nmbrOfWlkrs / 2, 0, 1 );

  int sti = 0, sbi;
  while ( sti < chn[0].nmbrOfStps ) {
    sbi = 0;
    while ( sbi < 2 ) {
      Propose ( sti, sbi, chn );
      Priors ( chn[0].nmbrOfWlkrs / 2, chn[0].prpsdWlkrs, chn[0].prpsdPrrs );
      Statistics ( chn[0].nmbrOfWlkrs / 2, chn[0].prpsdWlkrs, chn[0].prpsdSttstcs );
      Update ( sti, sbi, chn );
      sbi += 1;
    }
    ToChain ( sti, chn );
    sti += 1;
  }
  printf ( "      ... >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>> Done!\n" );

  hipEventRecord ( cdp[0].stop, 0 );
  hipEventSynchronize ( cdp[0].stop );
  hipEventElapsedTime ( &chn[0].elapsedTime, cdp[0].start, cdp[0].stop );

  hipEventRecord ( cdp[0].start, 0 );

  /* Autocorrelation function */
  int NN[RANK] = { chn[0].nmbrOfStps };
  cdp[0].cufftRes = hipfftPlanMany ( &cdp[0].cufftPlan, RANK, NN, NULL, 1, chn[0].nmbrOfStps, NULL, 1, chn[0].nmbrOfStps, HIPFFT_C2C, chn[0].nmbrOfWlkrs );
  ReturnChainFunction <<< Grid ( chn[0].nmbrOfWlkrs, chn[0].nmbrOfStps ), dimBlock >>> ( chn[0].nmbrOfStps, chn[0].nmbrOfWlkrs, 0, chn[0].chnOfWlkrs, chn[0].chnFnctn );
  AutocorrelationFunctionAveraged ( cdp[0].cufftRes, cdp[0].cublasStat, cdp[0].cublasHandle, cdp[0].cufftPlan, chn[0].nmbrOfStps, chn[0].nmbrOfWlkrs, chn[0].chnFnctn, chn[0].atCrrFnctn );

  hipEventRecord ( cdp[0].stop, 0 );
  hipEventSynchronize ( cdp[0].stop );
  hipEventElapsedTime ( &chn[0].cufftElapsedTime, cdp[0].start, cdp[0].stop );

  /* Autocorreation time */
  CumulativeSumOfAutocorrelationFunction ( chn[0].nmbrOfStps, chn[0].atCrrFnctn, chn[0].cmSmAtCrrFnctn );
  int MM = ChooseWindow ( chn[0].nmbrOfStps, 5e0f, chn[0].cmSmAtCrrFnctn );
  chn[0].atcTime = 2 * chn[0].cmSmAtCrrFnctn[MM] - 1e0f;

  printf ( ".................................................................\n" );
  printf ( " Autocorrelation time window -- %i\n", MM );
  printf ( " Autocorrelation time -- %.8E\n", chn[0].atcTime );
  printf ( " Autocorrelation time threshold -- %.8E\n", chn[0].nmbrOfStps / 5e1f );
  printf ( " Effective number of independent samples -- %.8E\n", chn[0].nmbrOfWlkrs * chn[0].nmbrOfStps / chn[0].atcTime );
  printf ( ".................................................................\n" );
  printf ( " Time to generate: %3.1f ms\n", chn[0].elapsedTime );
  printf ( " Time to compute Autocorrelation Function: %3.1f ms\n", chn[0].cufftElapsedTime );
  printf ( "\n" );

  /* Write results to a file */
  SimpleWriteDataFloat ( "Autocor.out", chn[0].nmbrOfStps, chn[0].atCrrFnctn );
  SimpleWriteDataFloat ( "AutocorCM.out", chn[0].nmbrOfStps, chn[0].cmSmAtCrrFnctn );
  WriteChainToFile ( chn[0].thrdNm, chn[0].thrdIndx, chn[0].nmbrOfWlkrs, chn[0].nmbrOfStps, chn[0].chnOfWlkrs, chn[0].chnOfSttstcs, chn[0].chnOfPrrs );

  DestroyAllTheCudaStaff ( cdp );
  FreeChain ( chn );

  // Reset the device and exit
  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application exits

  cdp[0].err = hipDeviceReset ();
  if ( cdp[0].err != hipSuccess ) {
    fprintf ( stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString ( cdp[0].err ) );
    exit ( EXIT_FAILURE );
  }

  return 0;
}

#endif // _GWMCMCCUDA_CU_
