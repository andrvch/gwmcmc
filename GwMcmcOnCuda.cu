#include "hip/hip_runtime.h"
#ifndef _GWMCMCCUDA_CU_
#define _GWMCMCCUDA_CU_

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <fitsio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include "GwMcmcStructuresFunctionsAndKernels.cuh"

/* Functions and Kernels: */
__host__ __device__ int PriorCondition ( const Walker wlkr )
{
  int indx, cndtn = 1;
  indx = 0;
  cndtn = cndtn * ( wlkr.par[indx] > 0 ) * ( wlkr.par[indx] < 5.5 );
  indx = 1; // pl normalization
  cndtn = cndtn * ( wlkr.par[indx] > -15. );
  indx = 2; // Temperature
  cndtn = cndtn * ( wlkr.par[indx] > 0.03 ) * ( wlkr.par[indx] < 1. );
  indx = 3; // Radi
  //cndtn = cndtn * ( wlkr.par[indx] > 0.0 );
  indx = 4; // Distance
  cndtn = cndtn * ( wlkr.par[indx] < 3.3 ) * ( wlkr.par[indx] > 1. );
  indx = NHINDX; // Hydrogen column density
  cndtn = cndtn * ( wlkr.par[indx] > 0 );
  return cndtn;
}

__host__ __device__ float PriorStatistic ( const Walker wlkr, const int cndtn, const float mNh, const float sNh )
{
  int indx = NHINDX; // Hydrogen column density
  float prr = 0, sum = 0, mean = 0, sigma = 0.06;
  float theta = powf ( sNh, 2 ) / mNh;
  float kk = mNh / theta;
  sum = sum + ( kk - 1 ) * logf ( wlkr.par[indx] ) - wlkr.par[indx] / theta;
  //sum = sum + powf ( ( wlkr.par[indx] - mNh ) / sNh, 2 );
  indx = NHINDX + 1;
  while ( indx < NPRS )
  {
    sum = sum + powf ( ( wlkr.par[indx] - mean ) / sigma, 2 );
    indx += 1;
  }
  if ( cndtn ) { prr = sum; } else { prr = INF; }
  return prr;
}

__global__ void AssembleArrayOfAbsorptionFactors ( const int nmbrOfWlkrs, const int nmbrOfEnrgChnnls, const int nmbrOfElmnts,
                                                   const float *crssctns, const float *abndncs, const int *atmcNmbrs, const Walker *wlkrs, float *absrptnFctrs )
{
  int enIndx = threadIdx.x + blockDim.x * blockIdx.x;
  int wlIndx = threadIdx.y + blockDim.y * blockIdx.y;
  int ttIndx = enIndx + wlIndx * nmbrOfEnrgChnnls;
  int elIndx, effElIndx, crIndx, prIndx;
  float xsctn, clmn, nh;
  if ( ( enIndx < nmbrOfEnrgChnnls ) && ( wlIndx < nmbrOfWlkrs ) )
  {
    elIndx = 0;
    prIndx = elIndx + NHINDX;
    crIndx = elIndx + enIndx * nmbrOfElmnts;
    effElIndx = atmcNmbrs[elIndx] - 1;
    nh = wlkrs[wlIndx].par[prIndx] * 1.E22;
    clmn = abndncs[effElIndx];
    xsctn = clmn * crssctns[crIndx];
    elIndx = 1;
    while ( elIndx < nmbrOfElmnts )
    {
      prIndx = elIndx + NHINDX;
      crIndx = elIndx + enIndx * nmbrOfElmnts;
      effElIndx = atmcNmbrs[elIndx] - 1;
      clmn = abndncs[effElIndx]; // * powf ( 10, wlkrs[wlIndx].par[prIndx] );
      xsctn = xsctn + clmn * crssctns[crIndx];
      elIndx += 1;
    }
    absrptnFctrs[ttIndx] = expf ( - nh * xsctn );
  }
}

__global__ void AssembleArrayOfModelFluxes ( const int nmbrOfWlkrs, const int nmbrOfEnrgChnnls,
                                             const float *enrgChnnls, const float *arfFctrs, const float *absrptnFctrs, const Walker *wlkrs,
                                             float *mdlFlxs )
{
  int enIndx = threadIdx.x + blockDim.x * blockIdx.x;
  int wlIndx = threadIdx.y + blockDim.y * blockIdx.y;
  int ttIndx = enIndx + wlIndx * nmbrOfEnrgChnnls;
  if ( ( enIndx < nmbrOfEnrgChnnls ) && ( wlIndx < nmbrOfWlkrs ) )
  {
    //mdlFlxs[ttIndx] =  arfFctrs[enIndx] * absrptnFctrs[ttIndx] * BlackBody ( wlkrs[wlIndx].par[0], wlkrs[wlIndx].par[1], enrgChnnls[enIndx], enrgChnnls[enIndx+1] );
    //mdlFlxs[ttIndx] =  arfFctrs[enIndx] * absrptnFctrs[ttIndx] * PowerLaw ( wlkrs[wlIndx].par[0], wlkrs[wlIndx].par[1], enrgChnnls[enIndx], enrgChnnls[enIndx+1] );
    mdlFlxs[ttIndx] =  arfFctrs[enIndx] * absrptnFctrs[ttIndx] * ( PowerLaw ( wlkrs[wlIndx].par[0], wlkrs[wlIndx].par[1], enrgChnnls[enIndx], enrgChnnls[enIndx+1] ) + BlackBody ( wlkrs[wlIndx].par[2], wlkrs[wlIndx].par[3], enrgChnnls[enIndx], enrgChnnls[enIndx+1] ) );
  }
}

__host__ int ModelFluxes ( const float *abndncs, const int *atmcNmbrs, const int nmbrOfWlkrs, const Walker *wlkrs, const int nmbrOfEnrgChnnls, const float *crssctns, const float *enrgChnnls, const float *arfFctrs, float *absrptnFctrs, float *mdlFlxs )
{
  dim3 dimBlock ( THRDSPERBLCK, THRDSPERBLCK );
  dim3 dimGrid = Grid ( nmbrOfEnrgChnnls, nmbrOfWlkrs );
  AssembleArrayOfAbsorptionFactors <<< dimGrid, dimBlock >>> ( nmbrOfWlkrs, nmbrOfEnrgChnnls, ATNMR, crssctns, abndncs, atmcNmbrs, wlkrs, absrptnFctrs );
  /* 4 a ) Assemble array of nsa fluxes */
  //BilinearInterpolation <<< dimGrid_0, dimBlock >>> ( chn[0].nmbrOfWlkrs, spc[0].nmbrOfEnrgChnnls, 2, mdl[0].nsaFlxs, mdl[0].nsaE, mdl[0].nsaT, mdl[0].numNsaE, mdl[0].numNsaT, spc[0].enrgChnnls, chn[0].wlkrs, spc[0].mdlFlxs );
  /* 4 ) Assemble array of model fluxes, spc[0].mdlFlxs[chn[0].nmbrOfWlkrs*spc[0].nmbrOfEnrgChnnls] */
  AssembleArrayOfModelFluxes <<< dimGrid, dimBlock >>> ( nmbrOfWlkrs, nmbrOfEnrgChnnls, enrgChnnls, arfFctrs, absrptnFctrs, wlkrs, mdlFlxs );
  return 0;
}

__host__ int Priors ( const int nmbrOfDistBins, const float *Dist, const float *EBV, const float *errEBV, const int nmbrOfWlkrs, Walker *prpsdWlkrs, float *mNh, float *sNh, float *prrs )
{
  int blocksPerThread = Blocks ( nmbrOfWlkrs );
  LinearInterpolation <<< blocksPerThread, THRDSPERBLCK >>> ( nmbrOfWlkrs, nmbrOfDistBins, 4, Dist, EBV, errEBV, prpsdWlkrs, mNh, sNh );
  AssembleArrayOfPriors <<< blocksPerThread, THRDSPERBLCK >>> ( nmbrOfWlkrs, prpsdWlkrs, mNh, sNh, prrs );
  return 0;
}

/**
 * Host main routine
 */
int main ( int argc, char *argv[] )
{
  dim3 dimBlock ( THRDSPERBLCK, THRDSPERBLCK );
  const int verbose = 1;
  const float lwrNtcdEnrg = 0.3;
  const float hghrNtcdEnrg = 8.0;
  const float dlt = 1.E-4;
  const float phbsPwrlwInt[NPRS] = { 1.1, log10f ( 9.E-6 ), 0.1, -3., log10f ( 8E2 ), 0.15 };

  /* Initialize */
  Cuparam cdp[NSPCTR];
  Model mdl[NSPCTR];
  Chain chn[NSPCTR];
  Spectrum spc[NSPCTR];

  cdp[0].dev = atoi( argv[1] );
  const char *spcFl = argv[2];
  const char *spcLst[NSPCTR] = { spcFl, spcFl };
  chn[0].thrdNm = argv[3];
  chn[0].nmbrOfWlkrs = atoi ( argv[4] );
  chn[0].nmbrOfStps = atoi ( argv[5] );
  chn[0].thrdIndx = atoi ( argv[6] );
  chn[0].dlt = dlt;
  spc[0].lwrNtcdEnrg = lwrNtcdEnrg;
  spc[0].hghrNtcdEnrg = hghrNtcdEnrg;
  spc[1].lwrNtcdEnrg = lwrNtcdEnrg;
  spc[1].hghrNtcdEnrg = hghrNtcdEnrg;

  InitializeCuda ( cdp );
  InitializeModel ( mdl );
  InitializeChain ( cdp, phbsPwrlwInt, chn );
  InitializeSpectra ( spcLst, cdp, verbose, chn, mdl, spc );

  /* Initialize walkers */
  if ( chn[0].thrdIndx == 0 )
  {
    InitializeWalkers ( cdp, chn[0].nmbrOfWlkrs, chn[0].rndmVls, chn[0].dlt, chn[0].strtngWlkr, chn[0].wlkrs, chn[0].sttstcs );

    ModelFluxes ( mdl[0].abndncs, mdl[0].atmcNmbrs, chn[0].nmbrOfWlkrs, chn[0].wlkrs, spc[0].nmbrOfEnrgChnnls, spc[0].crssctns, spc[0].enrgChnnls, spc[0].arfFctrs, spc[0].absrptnFctrs, spc[0].mdlFlxs );
    FoldModelFluxes ( cdp, chn[0].nmbrOfWlkrs, spc[0].nmbrOfChnnls, spc[0].nmbrOfEnrgChnnls, spc[0].nmbrOfRmfVls, spc[0].rmfVls, spc[0].rmfPntr, spc[0].rmfIndx, spc[0].mdlFlxs, spc[0].flddMdlFlxs );
    Statistics ( chn[0].nmbrOfWlkrs, spc[0].nmbrOfChnnls, spc[0].srcExptm, spc[0].bckgrndExptm, spc[0].srcCnts, spc[0].bckgrndCnts, spc[0].flddMdlFlxs, spc[0].chnnlSttstcs );
    SumUpStatistics ( cdp, chn[0].nmbrOfWlkrs, spc[0].nmbrOfChnnls, spc[0].chnnlSttstcs, spc[0].ntcdChnnls, chn[0].sttstcs );
  }
  else if ( chn[0].thrdIndx > 0 )
  {
    InitializeWalkersAndStatistics ( chn[0].nmbrOfWlkrs, chn[0].lstWlkrsAndSttstcs, chn[0].wlkrs, chn[0].sttstcs );
  }

  hipEventRecord ( cdp[0].start, 0 );

  /* Run chain */
  printf ( ".................................................................\n" );
  printf ( " Start ...                                                  \n" );
  hiprandGenerateUniform ( cdp[0].curandGnrtr, chn[0].rndmVls, chn[0].nmbrOfRndmVls );
  int stpIndx = 0, sbstIndx;
  while ( stpIndx < chn[0].nmbrOfStps )
  {
    sbstIndx = 0;
    while ( sbstIndx < 2 )
    {
      Propose ( stpIndx, sbstIndx, chn[0].nmbrOfWlkrs / 2, chn[0].wlkrs, chn[0].rndmVls, chn[0].zRndmVls, chn[0].prpsdWlkrs, chn[0].prpsdSttstcs );

      Priors ( mdl[0].nmbrOfDistBins, mdl[0].Dist, mdl[0].EBV, mdl[0].errEBV, chn[0].nmbrOfWlkrs / 2, chn[0].prpsdWlkrs, chn[0].mNh, chn[0].sNh, chn[0].prrs );

      ModelFluxes ( mdl[0].abndncs, mdl[0].atmcNmbrs, chn[0].nmbrOfWlkrs / 2, chn[0].prpsdWlkrs, spc[0].nmbrOfEnrgChnnls, spc[0].crssctns, spc[0].enrgChnnls, spc[0].arfFctrs, spc[0].absrptnFctrs, spc[0].mdlFlxs );
      FoldModelFluxes ( cdp, chn[0].nmbrOfWlkrs / 2, spc[0].nmbrOfChnnls, spc[0].nmbrOfEnrgChnnls, spc[0].nmbrOfRmfVls, spc[0].rmfVls, spc[0].rmfPntr, spc[0].rmfIndx, spc[0].mdlFlxs, spc[0].flddMdlFlxs );
      Statistics ( chn[0].nmbrOfWlkrs / 2, spc[0].nmbrOfChnnls, spc[0].srcExptm, spc[0].bckgrndExptm, spc[0].srcCnts, spc[0].bckgrndCnts, spc[0].flddMdlFlxs, spc[0].chnnlSttstcs );
      SumUpStatistics ( cdp, chn[0].nmbrOfWlkrs / 2, spc[0].nmbrOfChnnls, spc[0].chnnlSttstcs, spc[0].ntcdChnnls, chn[0].prpsdSttstcs );

      Update ( stpIndx, sbstIndx, chn[0].nmbrOfWlkrs / 2, chn[0].prpsdWlkrs, chn[0].prpsdSttstcs, chn[0].prrs, chn[0].rndmVls, chn[0].zRndmVls, chn[0].wlkrs, chn[0].sttstcs );
      sbstIndx += 1;
    }
    WriteWalkersAndStatisticsToChain <<< Blocks ( chn[0].nmbrOfWlkrs ), THRDSPERBLCK >>> ( chn[0].nmbrOfWlkrs, stpIndx, chn[0].wlkrs, chn[0].sttstcs, chn[0].chnOfWlkrs, chn[0].chnOfSttstcs );
    stpIndx += 1;
  }
  printf ( "      ... >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>> Done!\n" );

  hipEventRecord ( cdp[0].stop, 0 );
  hipEventSynchronize ( cdp[0].stop );
  hipEventElapsedTime ( &chn[0].elapsedTime, cdp[0].start, cdp[0].stop );

  hipEventRecord ( cdp[0].start, 0 );

  /* Autocorrelation function */
  int NN[RANK] = { chn[0].nmbrOfStps };
  cdp[0].cufftRes = hipfftPlanMany ( &cdp[0].cufftPlan, RANK, NN, NULL, 1, chn[0].nmbrOfStps, NULL, 1, chn[0].nmbrOfStps, HIPFFT_C2C, chn[0].nmbrOfWlkrs );
  if ( cdp[0].cufftRes != HIPFFT_SUCCESS ) { fprintf ( stderr, "CUFFT error: Direct Plan configuration failed" ); return 1; }
  ReturnChainFunction <<< Grid ( chn[0].nmbrOfWlkrs, chn[0].nmbrOfStps ), dimBlock >>> ( chn[0].nmbrOfStps, chn[0].nmbrOfWlkrs, 0, chn[0].chnOfWlkrs, chn[0].chnFnctn );
  AutocorrelationFunctionAveraged ( cdp[0].cufftRes, cdp[0].cublasStat, cdp[0].cublasHandle, cdp[0].cufftPlan, chn[0].nmbrOfStps, chn[0].nmbrOfWlkrs, chn[0].chnFnctn, chn[0].atCrrFnctn );

  hipEventRecord ( cdp[0].stop, 0 );
  hipEventSynchronize ( cdp[0].stop );
  hipEventElapsedTime ( &chn[0].cufftElapsedTime, cdp[0].start, cdp[0].stop );

  /* Autocorreation time */
  CumulativeSumOfAutocorrelationFunction ( chn[0].nmbrOfStps, chn[0].atCrrFnctn, chn[0].cmSmAtCrrFnctn );
  int MM = ChooseWindow ( chn[0].nmbrOfStps, 5e0f, chn[0].cmSmAtCrrFnctn );
  chn[0].atcTime = 2 * chn[0].cmSmAtCrrFnctn[MM] - 1e0f;
  printf ( ".................................................................\n" );
  printf ( " Autocorrelation time window -- %i\n", MM );
  printf ( " Autocorrelation time -- %.8E\n", chn[0].atcTime );
  printf ( " Autocorrelation time threshold -- %.8E\n", chn[0].nmbrOfStps / 5e1f );
  printf ( " Effective number of independent samples -- %.8E\n", chn[0].nmbrOfWlkrs * chn[0].nmbrOfStps / chn[0].atcTime );

  /* Elapsed time */
  printf ( ".................................................................\n" );
  printf ( " Time to generate: %3.1f ms\n", chn[0].elapsedTime );
  printf ( " Time to compute Autocorrelation Function: %3.1f ms\n", chn[0].cufftElapsedTime );
  printf ( "\n" );

  /* Write results to a file */
  SimpleWriteDataFloat ( "Autocor.dat", chn[0].nmbrOfStps, chn[0].atCrrFnctn );
  SimpleWriteDataFloat ( "AutocorCM.dat", chn[0].nmbrOfStps, chn[0].cmSmAtCrrFnctn );
  WriteChainToFile ( chn[0].thrdNm, chn[0].thrdIndx, chn[0].nmbrOfWlkrs, chn[0].nmbrOfStps, chn[0].chnOfWlkrs, chn[0].chnOfSttstcs );

  /* Destroy cuda related contexts and things: */
  DestroyAllTheCudaStaff ( cdp );

  /* Free memory: */
  FreeSpec ( spc );
  FreeChain ( chn );
  FreeModel ( mdl );

  // Reset the device and exit
  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application exits

  cdp[0].err = hipDeviceReset ( );
  if ( cdp[0].err != hipSuccess )
  {
    fprintf ( stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString ( cdp[0].err ) );
    exit ( EXIT_FAILURE );
  }

  return 0;
}

#endif // _GWMCMCCUDA_CU_
