#ifndef _GWMCMCCUDA_CU_
#define _GWMCMCCUDA_CU_

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <fitsio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include "StrctrsAndFnctns.cuh"

int main ( int argc, char *argv[] ) {
  const int vrb = 1;

  Cupar cdp[1];
  cdp[0].dev = atoi ( argv[1] );

  initializeCuda ( cdp );

  if ( vrb ) {
    printf ( "\n" );
    printf ( " Device ID: %d\n", cdp[0].dev );
    printf ( " Device name: %s\n", cdp[0].prop.name );
    printf ( " Driver API: v%d \n", cdp[0].driverVersion[0] );
    printf ( " Runtime API: v%d \n", cdp[0].runtimeVersion[0] );
    printf ( "\n" );
  }

  Chain chn[1];
  chn[0].name = argv[2];
  chn[0].nwl = atoi ( argv[3] );
  chn[0].nst = atoi ( argv[4] );
  chn[0].indx = atoi ( argv[5] );
  chn[0].ds = atoi ( argv[6] );
  chn[0].em = atoi ( argv[7] );
  chn[0].en = atoi ( argv[8] );
  chn[0].enn = chn[0].en * ( chn[0].en - 1 ) / 2;
  chn[0].dim = chn[0].ds * chn[0].em * chn[0].en;
  chn[0].dlt = 1.E-2;

  allocateChain ( chn );
  initializeChain ( cdp, chn );

  if ( vrb ) {
    printf ( " Start ... \n" );
  }

  hipEventRecord ( cdp[0].start, 0 );

  initializeRandomForStreach ( cdp, chn );
  //initializeRandomForWalk ( cdp, chn );
  //initializeRandomForMetropolis ( cdp, chn );

  chn[0].ist = 0;
  while ( chn[0].ist < chn[0].nst ) {
    /*
    metropolisMove ( cdp, chn );
    statisticMetropolis ( cdp, chn );
    metropolisUpdate ( cdp, chn );
    */
    chn[0].isb = 0;
    while ( chn[0].isb < 2 ) {
      //walkMove ( cdp, chn );
      streachMove ( cdp, chn );
      //metropolisMove ( cdp, chn );
      //hipDeviceSynchronize ();
      //printMetropolisMove ( chn );
      statistic ( cdp, chn );
      //statisticMetropolis ( cdp, chn );
      //hipDeviceSynchronize ();
      //printMetropolisMove ( chn );
      //printMove ( chn );
      //walkUpdate ( cdp, chn );
      streachUpdate ( cdp, chn );
      //metropolisUpdate ( cdp, chn );
      //hipDeviceSynchronize ();
      //printMetropolisUpdate ( chn );
      //printUpdate ( chn );
      chn[0].isb += 1;
    }
    saveCurrent ( chn );
    chn[0].ist += 1;
  }

  if ( vrb ) {
    printf ( " Done!\n" );
    printf ( "\n" );
  }

  hipEventRecord ( cdp[0].stop, 0 );
  hipEventSynchronize ( cdp[0].stop );
  hipEventElapsedTime ( &chn[0].time, cdp[0].start, cdp[0].stop );

  if ( vrb ) {
    printf ( " Time to generate: %3.1f ms\n", chn[0].time );
  }

  hipEventRecord ( cdp[0].start, 0 );

  averagedAutocorrelationFunction ( cdp, chn );

  hipEventRecord ( cdp[0].stop, 0 );
  hipEventSynchronize ( cdp[0].stop );
  hipEventElapsedTime ( &chn[0].time, cdp[0].start, cdp[0].stop );

  if ( vrb ) {
    printf ( " Autocorrelation time window: %i\n", chn[0].mmm );
    printf ( " Autocorrelation time: %.8E\n", chn[0].atcTime );
    printf ( " Autocorrelation time threshold: %.8E\n", chn[0].nst / 5e1f );
    printf ( " Effective number of independent samples: %.8E\n", chn[0].nwl * chn[0].nst / chn[0].atcTime );
    printf ( " Time to compute acor time: %3.1f ms\n", chn[0].time );
    printf ( "\n" );
  }

  /* Write results to a file */
  printf ( " Write results to the host memory and clean up ... \n" );
  simpleWriteDataFloat ( "Autocor.out", chn[0].nst, chn[0].atcrrFnctn );
  simpleWriteDataFloat ( "AutocorCM.out", chn[0].nst, chn[0].cmSmAtCrrFnctn );
  writeChainToFile ( chn[0].name, chn[0].indx, chn[0].dim, chn[0].nwl, chn[0].nst, chn[0].smpls, chn[0].stat );

  destroyCuda ( cdp );
  freeChain ( chn );

  // Reset the device and exit
  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application exits

  cdp[0].err = hipDeviceReset ();
  if ( cdp[0].err != hipSuccess ) {
    fprintf ( stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString ( cdp[0].err ) );
    exit ( EXIT_FAILURE );
  }

  printf ( " Done!\n" );

  return 0;
}

#endif // _GWMCMCCUDA_CU_
