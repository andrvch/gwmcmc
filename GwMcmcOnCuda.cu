#include "hip/hip_runtime.h"
#ifndef _GWMCMCCUDA_CU_
#define _GWMCMCCUDA_CU_

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <fitsio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include "StrctrsAndFnctns.cuh"

int main ( int argc, char *argv[] ) {
  const int vrb = 1;

  Cupar cdp[1];
  cdp[0].dev = atoi ( argv[1] );

  initializeCuda ( cdp );

  if ( vrb ) {
    printf ( "\n" );
    printf ( ".................................................................\n" );
    printf ( " CUDA device ID: %d\n", cdp[0].dev );
    printf ( " CUDA device Name: %s\n", cdp[0].prop.name );
    printf ( " Driver API: v%d \n", cdp[0].driverVersion[0] );
    printf ( " Runtime API: v%d \n", cdp[0].runtimeVersion[0] );
  }

  Chain chn[1];
  chn[0].dfl = argv[2];
  chn[0].name = argv[3];
  chn[0].nwl = atoi ( argv[4] );
  chn[0].nst = atoi ( argv[5] );
  chn[0].indx = atoi ( argv[6] );
  chn[0].dim = 2;
  chn[0].dlt = 1.E-6;
  chn[0].nbm = atoi ( argv[7] );

  readTimesInfo ( chn[0].dfl, &chn[0].nph, &chn[0].exptm );
  //chn[0].nph = 372383;
  //chn[0].nph = 272217;

  allocateChain ( chn );
  allocateTimes ( chn );

  readTimesData ( chn[0].dfl, chn[0].nph, chn[0].atms );

  simpleReadDataFloat ( chn[0].dfl, chn[0].atms );

  chn[0].scale = chn[0].nph * logf ( chn[0].nbm ) + chn[0].nph * logf ( chn[0].nph * 1. ) - logf ( chn[0].nph * 1. ) - ( chn[0].nph + chn[0].nbm - 1. ) * logf ( ( chn[0].nph + chn[0].nbm - 1. ) * 1. ) + logf ( ( chn[0].nph + chn[0].nbm - 1. ) * 1. );

  int sumsum = 0;
  for ( int i = 0; i < chn[0].nbm-2; i++ ) {
    sumsum += logf ( i + 2 );
  }
  chn[0].scale = chn[0].scale + sumsum;

  //for ( int i = 0; i < chn[0].dim; i++ ) {
  chn[0].x0[0] = ( float ) atof ( argv[8] ); //592.4215; //1.68799e-3; //3.362332;
  chn[0].x0[1] = 1./chn[0].nbm/2.;
  //}

  //for ( int i = 0; i < chn[0].dim; i++ ) {
  chn[0].xbnd[0] = 2.62;
  chn[0].xbnd[1] = 2.70;
  chn[0].xbnd[2] = 0.0;
  chn[0].xbnd[3] = 1./chn[0].nbm;
  //}

  initializeChain ( cdp, chn );

  if ( vrb ) {
    printf ( ".................................................................\n" );
    printf ( " Start ...                                                  \n" );
  }

  chn[0].sigma[0] = ( float ) atof ( argv[9] );
  chn[0].sigma[1] = 1. / chn[0].nbm / 10.;

  //hipDeviceSynchronize ();
  //printMetropolisMove ( chn );

  hipEventRecord ( cdp[0].start, 0 );

  //initializeRandomForStreach ( cdp, chn );
  //initializeRandomForWalk ( cdp, chn );
  initializeRandomForMetropolis ( cdp, chn );

  chn[0].ist = 0;
  while ( chn[0].ist < chn[0].nst ) {
    /*metropolisMove ( cdp, chn );
    statisticMetropolis ( cdp, chn );
    metropolisUpdate ( cdp, chn );*/
    chn[0].isb = 0;
    while ( chn[0].isb < chn[0].dim ) {
      //walkMove ( cdp, chn );
      //streachMove ( cdp, chn );
      metropolisMove ( cdp, chn );
      //hipDeviceSynchronize ();
      //printMetropolisMove ( chn );
      //statistic ( cdp, chn );
      //statisticMetropolis ( cdp, chn );
      modelStatistic1 ( cdp, chn );
      //hipDeviceSynchronize ();
      //printMetropolisMove ( chn );
      //printMove ( chn );
      //walkUpdate ( cdp, chn );
      //streachUpdate ( cdp, chn );
      metropolisUpdate ( cdp, chn );
      //hipDeviceSynchronize ();
      //printMetropolisUpdate ( chn );
      //printUpdate ( chn );
      chn[0].isb += 1;
    }
    saveCurrent ( chn );
    chn[0].ist += 1;
  }

  if ( vrb ) {
    printf ( "      ... >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>> Done!\n" );
  }

  hipEventRecord ( cdp[0].stop, 0 );
  hipEventSynchronize ( cdp[0].stop );
  hipEventElapsedTime ( &chn[0].time, cdp[0].start, cdp[0].stop );

  if ( vrb ) {
    printf ( ".................................................................\n" );
    printf ( " Time to generate: %3.1f ms\n", chn[0].time );
    printf ( "\n" );
  }

  hipEventRecord ( cdp[0].start, 0 );

  averagedAutocorrelationFunction ( cdp, chn );

  hipEventRecord ( cdp[0].stop, 0 );
  hipEventSynchronize ( cdp[0].stop );
  hipEventElapsedTime ( &chn[0].time, cdp[0].start, cdp[0].stop );

  if ( vrb ) {
    printf ( ".................................................................\n" );
    printf ( " Autocorrelation time window -- %i\n", chn[0].mmm );
    printf ( " Autocorrelation time -- %.8E\n", chn[0].atcTime );
    printf ( " Autocorrelation time threshold -- %.8E\n", chn[0].nst / 5e1f );
    printf ( " Effective number of independent samples -- %.8E\n", chn[0].nwl * chn[0].nst / chn[0].atcTime );
    printf ( " Time to compute acor time: %3.1f ms\n", chn[0].time );
    printf ( "\n" );
  }

  /* Write results to a file */
  simpleWriteDataFloat ( "Autocor.out", chn[0].nst, chn[0].atcrrFnctn );
  simpleWriteDataFloat ( "AutocorCM.out", chn[0].nst, chn[0].cmSmAtCrrFnctn );
  writeChainToFile ( chn[0].name, chn[0].indx, chn[0].dim, chn[0].nwl, chn[0].nst, chn[0].nbm, chn[0].smpls, chn[0].stat, chn[0].numbers, chn[0].prior );

  destroyCuda ( cdp );
  freeChain ( chn );
  freeTimes ( chn );

  // Reset the device and exit
  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application exits

  cdp[0].err = hipDeviceReset ();
  if ( cdp[0].err != hipSuccess ) {
    fprintf ( stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString ( cdp[0].err ) );
    exit ( EXIT_FAILURE );
  }

  return 0;
}

#endif // _GWMCMCCUDA_CU_
