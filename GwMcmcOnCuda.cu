#ifndef _GWMCMCCUDA_CU_
#define _GWMCMCCUDA_CU_

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <fitsio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include "StrctrsAndFnctns.cuh"

int main ( int argc, char *argv[] ) {
  const int vrb = 1;

  Cupar cdp[1];
  cdp[0].dev = atoi ( argv[1] );

  initializeCuda ( cdp );

  if ( vrb ) {
    printf ( "\n" );
    printf ( " Device ID: %d\n", cdp[0].dev );
    printf ( " Device name: %s\n", cdp[0].prop.name );
    printf ( " Driver API: v%d \n", cdp[0].driverVersion[0] );
    printf ( " Runtime API: v%d \n", cdp[0].runtimeVersion[0] );
    printf ( "\n" );
  }

  Chain chn[1];
  chn[0].name = argv[2];
  chn[0].nwl = atoi ( argv[3] );
  chn[0].nst = atoi ( argv[4] );
  chn[0].indx = atoi ( argv[5] );
  chn[0].dlt = 0.2E-4;

  Image img[NIMG];
  float pixdim = atoi ( argv[6] );
  for ( int i = 0; i < NIMG/2; i ++ ) {
    img[i].imdim = pixdim;
    img[i].nx = pixdim;
    img[i].ny = pixdim;
    img[i].pix = 1.;
  }

  img[0].psffl = argv[7];
  img[0].datafl = argv[8];
  img[1].psffl = argv[9];
  img[1].datafl = argv[10];
  img[2].psffl = argv[11];
  img[2].datafl = argv[12];

  img[0].xref = 4147.5912;
  img[0].yref = 3955.0466;
  img[1].xref = 4017.1864;
  img[1].yref = 4183.6101;
  img[2].xref = 4313.9623;
  img[2].yref = 4284.8801;

  chn[0].dim = 3 * ( NIMG / 2 );
  printf ( " chain dimension = %i \n ", chn[0].dim );

  allocateChain ( chn );
  allocateImage ( chn, img );

  for ( int i = 0; i < NIMG/2; i++ ) {
    img[i].idx = i;
  }

  /*
  for ( int i = 0; i < chn[0].dim-1; i++ ) {
    chn[0].x0[i] = 0.;
  }*/

  for ( int i = 0; i < NIMG/2; i++ ) {
    chn[0].x0[3*i] = 0.;
    chn[0].x0[3*i+1] = 0.;
    chn[0].x0[3*i+2] = 0.04;
  }

  for ( int i = 0; i < NIMG/2; i++ ) {
    chn[0].x0bn[6*i] = -5.;
    chn[0].x0bn[6*i+1] = 5.;
    chn[0].x0bn[6*i+2] = -5.;
    chn[0].x0bn[6*i+3] = 5.;
    chn[0].x0bn[6*i+4] = 0.0;
    chn[0].x0bn[6*i+5] = 10000.;
  }

  for ( int i = 0; i < NIMG/2; i++ ) {
    simpleReadDataFloat ( img[i].psffl, img[i].psf );
    simpleReadDataFloat ( img[i].datafl, img[i].img );
  }

  /*
  printf ( "Input psf file:" );
  printf ( "\n" );
  printf ( "\n" );
  for ( int i = 0; i < img[0].nx*img[0].ny; i ++ ) {
    printf ( " %4.4f ", img[0].psf[i] );
  }
  printf ( "\n" );
  printf ( "\n" );
  */

  initializeChain ( cdp, chn, img );

  /*
  hipDeviceSynchronize ();

  printf ( "Initial walkers:" );
  printf ( "\n" );
  printf ( "\n" );
  for ( int j = 0; j < chn[0].nwl; j ++ ) {
    for ( int i = 0; i < chn[0].dim; i ++ ) {
      printf ( " %4.4f " , chn[0].xx[i+j*chn[0].dim] );
    }
    printf ( "\n" );
    printf ( "\n" );
  }
  printf ( "Initial shifted psf's:" );
  printf ( "\n" );
  printf ( "\n" );
  for ( int j = 0; j < chn[0].nwl; j ++ ) {
    printf ( " walk num %i :\n", j );
    for ( int i = 0; i < chn[0].nx*chn[0].ny; i ++ ) {
      printf ( " %4.4f " , chn[0].pp[i+j*chn[0].nx*chn[0].ny] );
      //printf ( " %i " , chn[0].ww[i+j*chn[0].nx*chn[0].ny] );
    }
    printf ( "\n" );
    printf ( "\n" );
  }
  printf ( "Initial stat:" );
  printf ( "\n" );
  printf ( "\n" );
  for ( int j = 0; j < chn[0].nwl; j ++ ) {
    printf ( " walk num %i :\n", j );
    for ( int i = 0; i < chn[0].nx*chn[0].ny; i ++ ) {
      printf ( " %4.4f " , chn[0].sstt[i+j*chn[0].nx*chn[0].ny] );
      //printf ( " %i " , chn[0].ww[i+j*chn[0].nx*chn[0].ny] );
    }
    printf ( "\n" );
    printf ( "\n" );
  }
  printf ( "Initial total stat:" );
  printf ( "\n" );
  printf ( "\n" );
  for ( int j = 0; j < chn[0].nwl; j ++ ) {
    printf ( " walk num %i :\n", j );
    //for ( int i = 0; i < chn[0].nx*chn[0].ny; i ++ ) {
    printf ( " %4.4f " , chn[0].stt[j] );
      //printf ( " %i " , chn[0].ww[i+j*chn[0].nx*chn[0].ny] );
    //}
    printf ( "\n" );
    printf ( "\n" );
  }*/

  if ( vrb ) {
    printf ( " Start ... \n" );
  }

  hipEventRecord ( cdp[0].start, 0 );

  initializeRandomForStreach ( cdp, chn );
  //initializeRandomForWalk ( cdp, chn );
  //initializeRandomForMetropolis ( cdp, chn );

  chn[0].ist = 0;
  while ( chn[0].ist < chn[0].nst ) {
    /*metropolisMove ( cdp, chn );
    statisticMetropolis ( cdp, chn );
    metropolisUpdate ( cdp, chn );*/
    chn[0].isb = 0;
    while ( chn[0].isb < 2 ) {
      //walkMove ( cdp, chn );
      streachMove ( cdp, chn );
      //metropolisMove ( cdp, chn );
      /*
      hipDeviceSynchronize ();
      printf ( "Proposed walkers:" );
      printf ( "\n" );
      printf ( "\n" );
      for ( int j = 0; j < chn[0].nwl/2; j ++ ) {
        for ( int i = 0; i < chn[0].dim; i ++ ) {
          printf ( " %4.4f " , chn[0].xx1[i+j*chn[0].dim] );
        }
        printf ( "\n" );
        printf ( "\n" );
      }
      printf ( "Proposed shifted psf's:" );
      printf ( "\n" );
      printf ( "\n" );
      for ( int j = 0; j < chn[0].nwl/2; j ++ ) {
        printf ( " walk num %i :\n", j );
        for ( int i = 0; i < chn[0].nx*chn[0].ny; i ++ ) {
          printf ( " %4.4f " , chn[0].pp[i+j*chn[0].nx*chn[0].ny] );
          //printf ( " %i " , chn[0].ww[i+j*chn[0].nx*chn[0].ny] );
        }
        printf ( "\n" );
        printf ( "\n" );
      }*/
      //printMetropolisMove ( chn );
      statistic ( cdp, chn, img );
      //statisticMetropolis ( cdp, chn );
      /*
      hipDeviceSynchronize ();
      printf ( "Proposed stat:" );
      printf ( "\n" );
      printf ( "\n" );
      for ( int j = 0; j < chn[0].nwl/2; j ++ ) {
        printf ( " walk num %i :\n", j );
        for ( int i = 0; i < chn[0].nx*chn[0].ny; i ++ ) {
          printf ( " %4.4f " , chn[0].sstt1[i+j*chn[0].nx*chn[0].ny] );
          //printf ( " %i " , chn[0].ww[i+j*chn[0].nx*chn[0].ny] );
        }
        printf ( "\n" );
        printf ( "\n" );
      }
      //printMetropolisMove ( chn );
      //printMove ( chn );
      //walkUpdate ( cdp, chn );
      */
      streachUpdate ( cdp, chn );
      //metropolisUpdate ( cdp, chn );
      /*
      hipDeviceSynchronize ();
      printf ( "Updated walkers:" );
      printf ( "\n" );
      printf ( "\n" );
      for ( int j = 0; j < chn[0].nwl; j ++ ) {
        for ( int i = 0; i < chn[0].dim; i ++ ) {
          printf ( " %4.4f " , chn[0].xx[i+j*chn[0].dim] );
        }
        printf ( "\n" );
        printf ( "\n" );
      }
      printf ( "Updated stat:" );
      printf ( "\n" );
      printf ( "\n" );
      for ( int j = 0; j < chn[0].nwl; j ++ ) {
        printf ( " walk num %i :\n", j );
        //for ( int i = 0; i < chn[0].nx*chn[0].ny; i ++ ) {
        printf ( " %4.4f " , chn[0].stt[j] );
          //printf ( " %i " , chn[0].ww[i+j*chn[0].nx*chn[0].ny] );
        //}
        printf ( "\n" );
        printf ( "\n" );
      } */
      //printMetropolisUpdate ( chn );
      //printUpdate ( chn );
      chn[0].isb += 1;
    }
    saveCurrent ( chn );
    chn[0].ist += 1;
  }

  if ( vrb ) {
    printf ( " Done!\n" );
    printf ( "\n" );
  }

  hipEventRecord ( cdp[0].stop, 0 );
  hipEventSynchronize ( cdp[0].stop );
  hipEventElapsedTime ( &chn[0].time, cdp[0].start, cdp[0].stop );

  if ( vrb ) {
    printf ( " Time to generate: %3.1f ms\n", chn[0].time );
  }

  hipEventRecord ( cdp[0].start, 0 );

  averagedAutocorrelationFunction ( cdp, chn );

  hipEventRecord ( cdp[0].stop, 0 );
  hipEventSynchronize ( cdp[0].stop );
  hipEventElapsedTime ( &chn[0].time, cdp[0].start, cdp[0].stop );

  if ( vrb ) {
    printf ( " Autocorrelation time window: %i\n", chn[0].mmm );
    printf ( " Autocorrelation time: %.8E\n", chn[0].atcTime );
    printf ( " Autocorrelation time threshold: %.8E\n", chn[0].nwl * chn[0].nst / 5e1f );
    printf ( " Effective number of independent samples: %.8E\n", chn[0].nwl * chn[0].nst / chn[0].atcTime );
    printf ( " Time to compute acor time: %3.1f ms\n", chn[0].time );
    printf ( "\n" );
  }

  /* Write results to a file */
  printf ( " Write results to the host memory and clean up ... \n" );
  simpleWriteDataFloat ( "Autocor.out", chn[0].nst, chn[0].atcrrFnctn );
  simpleWriteDataFloat ( "AutocorCM.out", chn[0].nst, chn[0].cmSmAtCrrFnctn );
  writeChainToFile ( chn[0].name, chn[0].indx, chn[0].dim, chn[0].nwl, chn[0].nst, chn[0].smpls, chn[0].stat );

  destroyCuda ( cdp );
  freeChain ( chn );
  freeImage ( img );

  // Reset the device and exit
  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application exits

  cdp[0].err = hipDeviceReset ();
  if ( cdp[0].err != hipSuccess ) {
    fprintf ( stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString ( cdp[0].err ) );
    exit ( EXIT_FAILURE );
  }

  printf ( " Done!\n" );

  return 0;
}

#endif // _GWMCMCCUDA_CU_
