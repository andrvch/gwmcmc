#ifndef _GWMCMCCUDA_CU_
#define _GWMCMCCUDA_CU_

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <fitsio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include "StrctrsAndFnctns.cuh"

int main ( int argc, char *argv[] ) {
  const int vrb = 1;

  Cupar cdp[1];
  cdp[0].dev = atoi ( argv[1] );

  initializeCuda ( cdp );

  if ( vrb ) {
    printf ( "\n" );
    printf ( ".................................................................\n" );
    printf ( " CUDA device ID: %d\n", cdp[0].dev );
    printf ( " CUDA device Name: %s\n", cdp[0].prop.name );
    printf ( " Driver API: v%d \n", cdp[0].driverVersion[0] );
    printf ( " Runtime API: v%d \n", cdp[0].runtimeVersion[0] );
  }

  Chain chn[1];
  const char *spcFl1 = argv[2];
  const char *spcFl2 = argv[3];
  const char *spcFl3 = argv[4];
  const char *spcFl4 = argv[5];
  const char *spcFl5 = argv[6];
  const char *spcFl6 = argv[7];
  const char *spcFl7 = argv[8];
  const char *spcFl8 = argv[9];
  const char *spcFl9 = argv[10];
  const char *spcFl10 = argv[11];
  const char *spcFl11 = argv[12];
  const char *spcFl12 = argv[13];
  const char *spcLst[NSPCTR11] = { spcFl1, spcFl2, spcFl3, spcFl4, spcFl5, spcFl6, spcFl7, spcFl8, spcFl9, spcFl10, spcFl11, spcFl12 };

  chn[0].name = argv[NSPCTR11+2];
  chn[0].nwl = atoi ( argv[NSPCTR11+3] );
  chn[0].nst = atoi ( argv[NSPCTR11+4] );
  chn[0].indx = atoi ( argv[NSPCTR11+5] );
  chn[0].dim = NPRS;
  chn[0].dlt = 1.E-4;

  Model mdl[1];
  Spectrum spc[NSPCTR];

  const float lwrNtcdEnrg1 = ( float ) atof ( argv[NSPCTR11+6] );
  const float hghrNtcdEnrg1 = ( float ) atof ( argv[NSPCTR11+7] );

  for ( int i = 0; i < NSPCTR; i++ ) {
    spc[i].lwrNtcdEnrg = lwrNtcdEnrg1;
    spc[i].hghrNtcdEnrg = hghrNtcdEnrg1;
  }

  InitializeModel ( mdl );

  SpecInfo ( spcLst, vrb, spc );
  SpecAlloc ( chn, spc );
  SpecData ( cdp, vrb, mdl, spc );

  /*
  printf ( " Grouping Information -- \n " );
  int count = 0;
  for ( int i = 0; i < spc[0].nmbrOfBns; i++ ) {
    count += spc[0].grpPntr[i+1] - spc[0].grpPntr[i];
    printf ( " %i ", i );
    printf ( " %i ", spc[0].grpPntr[i+1] - spc[0].grpPntr[i] );
    for ( int j = spc[0].grpPntr[i]; j < spc[0].grpPntr[i+1]; j++ ) {
      printf ( " %2.0f ", spc[0].grpng[j] );
    }
    printf ( "\n" );
  }
  printf ( " Number of grouping bins -- %i\n ", count );
  printf ( " final bin -- %i\n", spc[0].grpPntr[spc[0].nmbrOfBns] - spc[0].grpPntr[spc[0].nmbrOfBns-1] );
  for ( int i = 0; i < spc[0].nmbrOfChnnls; i++ ) {
    printf ( " %i ", spc[0].grpIndx[i] );
  }
  printf ( "\n" );
  for ( int i = 0; i < spc[0].nmbrOfChnnls; i++ ) {
    printf ( " %2.0f ", spc[0].grpVls[i] );
  }
  printf ( "\n" );

  printf ( " %i\n", spc[0].nmbrOfNtcdBns );
  printf ( " %i\n", spc[0].nmbrOfUsdBns );
  printf ( " %i\n", spc[0].lwrBn );
  printf ( " %i\n", spc[0].hghrBn );
  printf ( " %i\n", spc[0].nmbrOfgrpIgnVls );

  for ( int i = 0; i < spc[0].nmbrOfgrpIgnVls; i++ ) {
    printf ( " %2.0f ", spc[0].grpIgnVls[i] );
  }
  printf ( " \n " );

  for ( int i = 0; i < spc[0].nmbrOfgrpIgnVls; i++ ) {
    printf ( " %i ", spc[0].grpIgnIndx[i] );
  }
  printf ( " \n " );

  for ( int i = 0; i < spc[0].nmbrOfNtcdBns+1; i++ ) {
    printf ( " %i ", spc[0].grpIgnPntr[i] );
  }
  printf ( " \n " );

  for ( int j = 0; j < NSPCTR; j++ ) {
    for ( int i = 0; i < spc[j].nmbrOfUsdBns; i++ ) {
      printf ( " %2.0f ", spc[j].srcGrp[i] );
    }
    printf ( " \n " );
  }*/

  allocateChain ( chn );

  chn[0].x0[0] = 5.9;
  chn[0].xbnd[0] = 5.5;
  chn[0].xbnd[1] = 6.5;

  chn[0].x0[1] = 0.0;
  chn[0].xbnd[2] = -5.;
  chn[0].xbnd[3] = 5;

  chn[0].x0[2] = 0.0;
  chn[0].xbnd[4] = -5.;
  chn[0].xbnd[5] = 5.;

  chn[0].x0[3] = 0.0;
  chn[0].xbnd[6] = -5.;
  chn[0].xbnd[7] = 5.;

  chn[0].x0[4] = 1.5;
  chn[0].xbnd[8] = -25.;
  chn[0].xbnd[9] = 25.;

  chn[0].x0[5] = -5.;
  chn[0].xbnd[10] = -25.;
  chn[0].xbnd[11] = 25.;

  chn[0].x0[6] = 1.5;
  chn[0].xbnd[12] = -25.;
  chn[0].xbnd[13] = 25.;

  chn[0].x0[7] = -5.;
  chn[0].xbnd[14] = -25.;
  chn[0].xbnd[15] = 25.;

  chn[0].x0[8] = 1.5;
  chn[0].xbnd[16] = -25.;
  chn[0].xbnd[17] = 25.;

  chn[0].x0[9] = -5.;
  chn[0].xbnd[18] = -25.;
  chn[0].xbnd[19] = 25.;

  chn[0].x0[10] = 1.5;
  chn[0].xbnd[20] = -25.;
  chn[0].xbnd[21] = 25.;

  chn[0].x0[11] = -5.;
  chn[0].xbnd[22] = -25.;
  chn[0].xbnd[23] = 25.;

  chn[0].x0[12] = 1.5;
  chn[0].xbnd[24] = -25.;
  chn[0].xbnd[25] = 25.;

  chn[0].x0[13] = -5.;
  chn[0].xbnd[26] = -25.;
  chn[0].xbnd[27] = 25.;

  chn[0].x0[14] = 1.5;
  chn[0].xbnd[28] = -25.;
  chn[0].xbnd[29] = 25.;

  chn[0].x0[15] = -5.;
  chn[0].xbnd[30] = -25.;
  chn[0].xbnd[31] = 25.;

  chn[0].x0[16] = 1.5;
  chn[0].xbnd[32] = -25.;
  chn[0].xbnd[33] = 25.;

  chn[0].x0[17] = -5.;
  chn[0].xbnd[34] = -25.;
  chn[0].xbnd[35] = 25.;

  chn[0].x0[18] = 1.5;
  chn[0].xbnd[36] = -25.;
  chn[0].xbnd[37] = 25.;

  chn[0].x0[19] = -5.;
  chn[0].xbnd[38] = -25.;
  chn[0].xbnd[39] = 25.;

  chn[0].x0[20] = 0.2;
  chn[0].xbnd[40] = 0.;
  chn[0].xbnd[41] = 25.;

  initializeChain ( cdp, chn, mdl, spc );

  if ( vrb ) {
    printf ( ".................................................................\n" );
    printf ( " Start ...                                                  \n" );
  }

  hipEventRecord ( cdp[0].start, 0 );

  initializeRandomForStreach ( cdp, chn );

  chn[0].ist = 0;
  while ( chn[0].ist < chn[0].nst ) {
    chn[0].isb = 0;
    while ( chn[0].isb < 2 ) {
      streachMove ( cdp, chn );
      modelStatistic1 ( cdp, mdl, chn, spc );
      streachUpdate ( cdp, chn, mdl );
      chn[0].isb += 1;
    }
    saveCurrent ( chn );
    chn[0].ist += 1;
  }

  if ( vrb ) {
    printf ( "      ... >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>> Done!\n" );
  }

  hipEventRecord ( cdp[0].stop, 0 );
  hipEventSynchronize ( cdp[0].stop );
  hipEventElapsedTime ( &chn[0].time, cdp[0].start, cdp[0].stop );

  if ( vrb ) {
    printf ( ".................................................................\n" );
    printf ( " Time to generate: %3.1f ms\n", chn[0].time );
    printf ( "\n" );
  }

  hipEventRecord ( cdp[0].start, 0 );

  averagedAutocorrelationFunction ( cdp, chn );

  hipEventRecord ( cdp[0].stop, 0 );
  hipEventSynchronize ( cdp[0].stop );
  hipEventElapsedTime ( &chn[0].time, cdp[0].start, cdp[0].stop );

  if ( vrb ) {
    printf ( ".................................................................\n" );
    printf ( " Autocorrelation time window -- %i\n", chn[0].mmm );
    printf ( " Autocorrelation time -- %.8E\n", chn[0].atcTime );
    printf ( " Autocorrelation time threshold -- %.8E\n", chn[0].nst / 5e1f );
    printf ( " Effective number of independent samples -- %.8E\n", chn[0].nwl * chn[0].nst / chn[0].atcTime );
    printf ( " Time to compute acor time: %3.1f ms\n", chn[0].time );
    printf ( "\n" );
  }

  /* Write results to a file */
  simpleWriteDataFloat ( "Autocor.out", chn[0].nst, chn[0].atcrrFnctn );
  simpleWriteDataFloat ( "AutocorCM.out", chn[0].nst, chn[0].cmSmAtCrrFnctn );
  writeChainToFile ( chn[0].name, chn[0].indx, chn[0].dim, chn[0].nwl, chn[0].nst, chn[0].smpls, chn[0].stat, chn[0].priors, chn[0].dist, chn[0].chiTwo );

  destroyCuda ( cdp );
  freeChain ( chn );
  FreeModel ( mdl );
  FreeSpec ( spc );

  // Reset the device and exit
  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application exits

  cdp[0].err = hipDeviceReset ();
  if ( cdp[0].err != hipSuccess ) {
    fprintf ( stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString ( cdp[0].err ) );
    exit ( EXIT_FAILURE );
  }

  return 0;
}

#endif // _GWMCMCCUDA_CU_
