#ifndef _GWMCMCCUDA_CU_
#define _GWMCMCCUDA_CU_

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <fitsio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include "StrctrsAndFnctns.cuh"

int main ( int argc, char *argv[] ) {
  const int vrb = 1;

  Cupar cdp[1];
  cdp[0].dev = atoi ( argv[1] );

  initializeCuda ( cdp );

  if ( vrb ) {
    printf ( "\n" );
    printf ( ".................................................................\n" );
    printf ( " CUDA device ID: %d\n", cdp[0].dev );
    printf ( " CUDA device Name: %s\n", cdp[0].prop.name );
    printf ( " Driver API: v%d \n", cdp[0].driverVersion[0] );
    printf ( " Runtime API: v%d \n", cdp[0].runtimeVersion[0] );
  }

  Chain chn[1];

  Spectrum spc[NSPCTR];
  Spectrum bkg[NSPCTR];
  for ( int i = 0; i < NSPCTR; i++ ) {
    spc[i].name = argv[2+2*i];
    bkg[i].name = argv[2+2*i+1];
  }

  chn[0].name = argv[NSPCTR11+2];
  chn[0].nwl = atoi ( argv[NSPCTR11+3] );
  chn[0].nst = atoi ( argv[NSPCTR11+4] );
  chn[0].indx = atoi ( argv[NSPCTR11+5] );
  chn[0].dim = NPRS;
  chn[0].dim1 = chn[0].dim + 3;
  chn[0].dlt = 1.E-4;
  chn[0].nkb = 100;

  for ( int i = 0; i < NSPCTR; i++ ) {
    spc[i].lwrNtcdEnrg = ( float ) atof ( argv[NSPCTR11+6] );
    spc[i].hghrNtcdEnrg = ( float ) atof ( argv[NSPCTR11+7] );
  }

  Model mdl[1];
  InitializeModel ( mdl );


  printf ( "\n" );
  for ( int i = 0; i < mdl[0].numCarbE; i++ ) {
    printf ( " %.8E\n ", mdl[0].carbE[i] );
  }
  printf ( "\n" );
  for ( int i = 0; i < mdl[0].numCarbG; i++ ) {
    printf ( " %.8E\n ", mdl[0].carbG[i] );
  }
  printf ( "\n" );
  printf ( "\n" );
  for ( int i = 0; i < mdl[0].numCarbT; i++ ) {
    printf ( " %.8E\n ", mdl[0].carbT[i] );
  }
  printf ( "\n" );


  SpecInfo ( vrb, spc );
  SpecInfo ( vrb, bkg );

  for ( int i = 0; i < NSPCTR; i++ ) {
    bkg[i].nmbrOfBns = spc[i].nmbrOfBns;
  }

  SpecAlloc ( chn, spc );
  SpecAlloc ( chn, bkg );

  SpecData ( cdp, vrb, mdl, spc, bkg );

  allocateChain ( chn );

  chn[0].x0[0] = 5.9;
  chn[0].xbnd[0] = 5.5;
  chn[0].xbnd[1] = 6.5;

  chn[0].x0[1] = 0.0;
  chn[0].xbnd[2] = -25.;
  chn[0].xbnd[3] = 25.;

  chn[0].x0[2] = 1.5;
  chn[0].xbnd[4] = -25.;
  chn[0].xbnd[5] = 25.;

  chn[0].x0[3] = -5.;
  chn[0].xbnd[6] = -25.;
  chn[0].xbnd[7] = 25.;

  chn[0].x0[4] = 0.1;
  chn[0].xbnd[8] = 0.0;
  chn[0].xbnd[9] = 25.;
/*
  chn[0].x0[7] = -5.;
  chn[0].xbnd[14] = -25.;
  chn[0].xbnd[15] = 25.;

  chn[0].x0[8] = 1.5;
  chn[0].xbnd[16] = -25.;
  chn[0].xbnd[17] = 25.;

  chn[0].x0[9] = -5.;
  chn[0].xbnd[18] = -25.;
  chn[0].xbnd[19] = 25.;

  chn[0].x0[10] = 1.2;
  chn[0].xbnd[20] = 1.0;
  chn[0].xbnd[21] = 2.5;
/*
  chn[0].x0[11] = 0.03;
  chn[0].xbnd[22] = 0.001;
  chn[0].xbnd[23] = 1.5;

  chn[0].x0[12] = 0.07;
  chn[0].xbnd[24] = 0.0;
  chn[0].xbnd[25] = 10.;

  chn[0].x0[13] = 0.1;
  chn[0].xbnd[26] = 0.;
  chn[0].xbnd[27] = 25.;
*/
  initializeChain ( cdp, chn );
  if ( chn[0].indx == 0 ) {
    modelStatistic0 ( cdp, mdl, chn, spc, bkg );
  }

  if ( vrb ) {
    printf ( ".................................................................\n" );
    printf ( " Start ...                                                  \n" );
  }

  hipEventRecord ( cdp[0].start, 0 );

  initializeRandomForStreach ( cdp, chn );
  chn[0].ist = 0;
  while ( chn[0].ist < chn[0].nst ) {
    chn[0].isb = 0;
    while ( chn[0].isb < 2 ) {
      streachMove ( cdp, chn );
      modelStatistic1 ( cdp, mdl, chn, spc, bkg );
      streachUpdate ( cdp, chn, mdl );
      chn[0].isb += 1;
    }
    saveCurrent ( chn );
    chn[0].ist += 1;
  }

  chainMomentsAndKde ( cdp, chn );

  if ( vrb ) {
    printf ( " Done!\n" );
  }

  hipEventRecord ( cdp[0].stop, 0 );
  hipEventSynchronize ( cdp[0].stop );
  hipEventElapsedTime ( &chn[0].time, cdp[0].start, cdp[0].stop );

  if ( vrb ) {
    printf ( ".................................................................\n" );
    printf ( " Time to generate -- %3.1f ms\n", chn[0].time );
  }

  hipEventRecord ( cdp[0].start, 0 );

  averagedAutocorrelationFunction ( cdp, chn );

  hipEventRecord ( cdp[0].stop, 0 );
  hipEventSynchronize ( cdp[0].stop );
  hipEventElapsedTime ( &chn[0].time, cdp[0].start, cdp[0].stop );

  if ( vrb ) {
    printf ( ".................................................................\n" );
    printf ( " Autocorrelation time window             -- %i\n", chn[0].mmm );
    printf ( " Autocorrelation time                    -- %.8E\n", chn[0].atcTime );
    printf ( " Autocorrelation time threshold          -- %.8E\n", chn[0].nwl * chn[0].nst / 5e1f );
    printf ( " Effective number of independent samples -- %.8E\n", chn[0].nwl * chn[0].nst / chn[0].atcTime );
    printf ( " Time to compute acor time               -- %3.1f ms\n", chn[0].time );
  }

  sortQKde ( chn );

  if ( vrb ) {
    printf ( ".................................................................\n" );
    printf ( " Medium                    -- " );
    for ( int i = 0; i < chn[0].dim1; i++ ) {
      printf ( " %2.2f ", chn[0].msmp[i] );
    }
    printf ( "\n" );
    printf ( " Std. deviation            -- " );
    for ( int i = 0; i < chn[0].dim1; i++ ) {
      printf ( " %2.2f ", chn[0].stdsmp[i] );
    }
    printf ( "\n" );
    printf ( " Max pdf (best-fit) values -- " );
    for ( int j = 0; j < chn[0].dim1; j++ ) {
        printf ( " %2.2f ", chn[0].skbin[j+0*chn[0].dim1] );
    }
    printf ( "\n" );
  }

  for ( int i = 0; i < chn[0].dim; i++ ) {
    chn[0].xx[i] = chn[0].skbin[i];
    printf ( " %2.2f ", chn[0].xx[i] );
  }
  //chn[0].xx[10] = 0.05;
  //chn[0].xx[11] = 100.;
  chn[0].didi[0] = chn[0].skbin[chn[0].dim];
  printf ( " %2.2f ", chn[0].didi[0] );
  chn[0].stt[0] = chn[0].skbin[chn[0].dim+1];
  printf ( " %2.2f ", chn[0].stt[0] );
  chn[0].chi[0] = chn[0].skbin[chn[0].dim+2];
  printf ( " %2.2f ", chn[0].chi[0] );
  printf ( "\n" );

  modelStatistic00 ( cdp, mdl, chn, spc, bkg );

  hipDeviceSynchronize ();

  /* Write results to a file */
  simpleWriteDataFloat ( "Autocor.out", chn[0].nst, chn[0].atcrrFnctn );
  simpleWriteDataFloat ( "AutocorCM.out", chn[0].nst, chn[0].cmSmAtCrrFnctn );
  writeSpectraToFile ( chn[0].name, spc, bkg );
  writeKdeToFile ( chn[0].name, chn[0].dim1, chn[0].nkb, chn[0].kbin, chn[0].kdePdf );
  writeWhalesToFile ( chn[0].name, chn[0].indx, chn[0].dim1, chn[0].nwl*chn[0].nst, chn[0].whales );
  writeChainToFile ( chn[0].name, chn[0].indx, chn[0].dim, chn[0].nwl, chn[0].nst, chn[0].smpls, chn[0].stat, chn[0].priors, chn[0].dist, chn[0].chiTwo );

  destroyCuda ( cdp );
  freeChain ( chn );
  FreeModel ( mdl );
  FreeSpec ( spc );
  FreeSpec ( bkg );

  // Reset the device and exit
  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application exits

  cdp[0].err = hipDeviceReset ();
  if ( cdp[0].err != hipSuccess ) {
    fprintf ( stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString ( cdp[0].err ) );
    exit ( EXIT_FAILURE );
  }

  return 0;
}

#endif // _GWMCMCCUDA_CU_
