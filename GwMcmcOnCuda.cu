#include "hip/hip_runtime.h"
#ifndef _GWMCMCCUDA_CU_
#define _GWMCMCCUDA_CU_

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <fitsio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include "StrctrsAndFnctns.cuh"

/* Functions and Kernels: */
__host__ __device__ int PriorCondition ( const Walker wlkr )
{
  int cndtn = 1;
  cndtn = cndtn * ( 5.5 < wlkr.par[TINDX] ) * ( wlkr.par[TINDX] < 6.5 );
<<<<<<< HEAD
  cndtn = cndtn * ( log10f ( 8. ) < wlkr.par[RINDX1] ) * ( wlkr.par[RINDX1] < log10f ( 20. ) );
  cndtn = cndtn * ( log10f ( 80. ) < wlkr.par[DINDX1] ) * ( wlkr.par[DINDX1] < log10f ( 2200. ) );
  cndtn = cndtn * ( log10f ( 0.7 ) < wlkr.par[3] ) * ( wlkr.par[3] < log10f ( 0.9 ) );
  cndtn = cndtn * ( log10f ( 0.01 ) < wlkr.par[4] ) * ( wlkr.par[4] < log10f ( 0.5 ) );
=======
>>>>>>> two_spectra+background
  cndtn = cndtn * ( 0. < wlkr.par[NHINDX] );
  return cndtn;
}

__host__ __device__ float PriorStatistic ( const Walker wlkr, const int cndtn, const float mNh1, const float sNh1, const float mNh2, const float sNh2 )
{
  float prr = 0, sum = 0;
  float theta = powf ( sNh1, 2 ) / mNh1;
  float kk = mNh1 / theta;
  //sum = sum + ( kk - 1 ) * logf ( wlkr.par[NHINDX] ) - wlkr.par[NHINDX] / theta;
  //sum = sum + powf ( ( wlkr.par[NHINDX] - mNh1 ) / sNh1, 2 );
  if ( cndtn ) { prr = sum; } else { prr = INF; }
  return prr;
}

__global__ void AssembleArrayOfModelFluxes ( const int spIndx, const int nmbrOfWlkrs, const int nmbrOfEnrgChnnls, const float backscal_src, const float backscal_bkg, const float *en, const float *arf, const float *absrptn, const Walker *wlk, const float *nsa1Flx, const float *nsa2Flx, float *flx )
{
  int e = threadIdx.x + blockDim.x * blockIdx.x;
  int w = threadIdx.y + blockDim.y * blockIdx.y;
  int t = e + w * nmbrOfEnrgChnnls;
  float f = 0, NormD, intNsaFlx;
  float scl = backscal_src / backscal_bkg;
  if ( ( e < nmbrOfEnrgChnnls ) && ( w < nmbrOfWlkrs ) )
  {
    if ( spIndx == 0 )
    {
<<<<<<< HEAD
      f = f + nsa1Flx[t]; // * powf ( 10., LOGPLANCK - log10f ( en[e+1] ) );
      f = f * GaussianAbsorption ( wlk[w].par[3], wlk[w].par[4], wlk[w].par[5], en[e+1] );
      f = f + PowerLaw ( wlk[w].par[6], wlk[w].par[7], en[e], en[e+1] );
      f = f * absrptn[t];
      f = f + scl * PowerLaw ( wlk[w].par[8], wlk[w].par[9], en[e], en[e+1] );
=======
      NormD = powf ( 10., - 2. * ( wlk[w].par[DINDX1] - KMCMPCCM ) );
      intNsaFlx = IntegrateNsa ( nsa1Flx[e+w*(nmbrOfEnrgChnnls+1)], nsa1Flx[e+1+w*(nmbrOfEnrgChnnls+1)], en[e], en[e+1] );
      f = f + NormD * intNsaFlx;
      f = f + PowerLaw ( wlk[w].par[2], wlk[w].par[3], en[e], en[e+1] );
      f = f * absrptn[t];
      f = f + scl * PowerLaw ( wlk[w].par[4], wlk[w].par[5], en[e], en[e+1] );
>>>>>>> two_spectra+background
      flx[t] = f * arf[e];
    }
    if ( spIndx == 1 )
    {
<<<<<<< HEAD
      f = f + PowerLaw ( wlk[w].par[8], wlk[w].par[9], en[e], en[e+1] );
=======
      f = f + PowerLaw ( wlk[w].par[4], wlk[w].par[5], en[e], en[e+1] );
      flx[t] = f * arf[e];
    }
    if ( spIndx == 2 )
    {
      NormD = powf ( 10., - 2. * ( wlk[w].par[DINDX1] - KMCMPCCM ) );
      intNsaFlx = NormD * IntegrateNsa ( nsa1Flx[e+w*(nmbrOfEnrgChnnls+1)], nsa1Flx[e+1+w*(nmbrOfEnrgChnnls+1)], en[e], en[e+1] );
      f = f + intNsaFlx;
      f = f + PowerLaw ( wlk[w].par[2], wlk[w].par[3], en[e], en[e+1] );
      f = f * absrptn[t];
      f = f + scl * PowerLaw ( wlk[w].par[6], wlk[w].par[7], en[e], en[e+1] );
      flx[t] = f * arf[e];
    }
    if ( spIndx == 3 )
    {
      f = f + PowerLaw ( wlk[w].par[6], wlk[w].par[7], en[e], en[e+1] );
>>>>>>> two_spectra+background
      flx[t] = f * arf[e];
    }
  }
}

__host__ int ModelFluxes ( const Model *mdl, const int nmbrOfWlkrs, const Walker *wlkrs, const int indx, Spectrum spec )
{
  dim3 dimBlock ( THRDSPERBLCK, THRDSPERBLCK );
  AssembleArrayOfAbsorptionFactors <<< Grid ( spec.nmbrOfEnrgChnnls, nmbrOfWlkrs ), dimBlock >>> ( nmbrOfWlkrs, spec.nmbrOfEnrgChnnls, ATNMR, spec.crssctns, mdl[0].abndncs, mdl[0].atmcNmbrs, wlkrs, spec.absrptnFctrs );
  //BilinearInterpolation <<< Grid ( spec.nmbrOfEnrgChnnls+1, nmbrOfWlkrs ), dimBlock >>> ( nmbrOfWlkrs, spec.nmbrOfEnrgChnnls, TINDX, RINDX1, DINDX1, mdl[0].nsmaxgFlxs, mdl[0].nsmaxgE, mdl[0].nsmaxgT, mdl[0].numNsmaxgE, mdl[0].numNsmaxgT, spec.enrgChnnls, wlkrs, spec.nsa1Flxs );
  BilinearInterpolation <<< Grid ( spec.nmbrOfEnrgChnnls+1, nmbrOfWlkrs ), dimBlock >>> ( nmbrOfWlkrs, spec.nmbrOfEnrgChnnls+1, TINDX, RINDX1, mdl[0].nsaFlxs, mdl[0].nsaE, mdl[0].nsaT, mdl[0].numNsaE, mdl[0].numNsaT, spec.enrgChnnls, wlkrs, spec.nsa1Flxs );
  AssembleArrayOfModelFluxes <<< Grid ( spec.nmbrOfEnrgChnnls, nmbrOfWlkrs ), dimBlock >>> ( indx, nmbrOfWlkrs, spec.nmbrOfEnrgChnnls, spec.backscal_src, spec.backscal_bkg, spec.enrgChnnls, spec.arfFctrs, spec.absrptnFctrs, wlkrs, spec.nsa1Flxs, spec.nsa2Flxs, spec.mdlFlxs );
  return 0;
}

__host__ int Priors ( const Model *mdl, Chain *chn )
{
  int blcks = Blocks ( chn[0].nmbrOfWlkrs / 2 );
  LinearInterpolation <<< blcks, THRDSPERBLCK >>> ( chn[0].nmbrOfWlkrs / 2, mdl[0].nmbrOfDistBins, DINDX1, mdl[0].Dist, mdl[0].EBV, mdl[0].errEBV, chn[0].prpsdWlkrs, chn[0].mNh1, chn[0].sNh1 );
  AssembleArrayOfPriors <<< blcks, THRDSPERBLCK >>> ( chn[0].nmbrOfWlkrs / 2, chn[0].prpsdWlkrs, chn[0].mNh1, chn[0].sNh1, chn[0].mNh2, chn[0].sNh2, chn[0].prrs );
  return 0;
}

/**
 * Host main routine
 */
int main ( int argc, char *argv[] )
{
  dim3 dimBlock ( THRDSPERBLCK, THRDSPERBLCK );
  const int verbose = 1;
  const float lwrNtcdEnrg = 0.5;
  const float hghrNtcdEnrg = 7.0;
  const float dlt = 1.E-4;
<<<<<<< HEAD
  const float phbsPwrlwInt[NPRS] = { 5.80, 1.0, 2.6, log10f ( 0.8 ), log10f ( 0.15 ), -1., 1.2, -5.2, 0.9, -5.0, 0.30 };
=======
  const float phbsPwrlwInt[NPRS] = { 6.0, 3.5, 1.0, -5.3, 0.90, -5.0, 1.2, -5.1, 0.17 };
>>>>>>> two_spectra+background

  /* Initialize */
  Cuparam cdp[NSPCTR];
  Model mdl[NSPCTR];
  Chain chn[NSPCTR];
  Spectrum spc[NSPCTR];

  cdp[0].dev = atoi( argv[1] );
  const char *spcFl1 = argv[2];
  const char *spcFl2 = argv[3];
  const char *spcFl3 = argv[4];
  const char *spcFl4 = argv[5];
  const char *spcLst[NSPCTR] = { spcFl1, spcFl2, spcFl3, spcFl4 };
  int NNspec = 4;
  chn[0].thrdNm = argv[NNspec+2];
  chn[0].nmbrOfWlkrs = atoi ( argv[NNspec+3] );
  chn[0].nmbrOfStps = atoi ( argv[NNspec+4] );
  chn[0].thrdIndx = atoi ( argv[NNspec+5] );
  chn[0].dlt = dlt;
  for ( int i = 0; i < NSPCTR; i++ )
  {
    spc[i].lwrNtcdEnrg = lwrNtcdEnrg;
    spc[i].hghrNtcdEnrg = hghrNtcdEnrg;
  }

  InitializeCuda ( cdp );
  InitializeModel ( mdl );
  InitializeChain ( cdp, phbsPwrlwInt, chn );

  SpecInfo ( spcLst, verbose, spc );
  SpecAlloc ( chn, spc );
  SpecData ( cdp, verbose, mdl, spc );

  /* Initialize walkers */
  if ( chn[0].thrdIndx == 0 )
  {
    InitAtRandom ( cdp, chn );
    for ( int i = 0; i < NSPCTR; i++ )
    {
      ModelFluxes ( mdl, chn[0].nmbrOfWlkrs, chn[0].wlkrs, i, spc[i] );
      FoldModel ( cdp, chn[0].nmbrOfWlkrs, spc[i] );
      Stat ( chn[0].nmbrOfWlkrs, spc[i] );
      SumUpStat ( cdp, 1, chn[0].nmbrOfWlkrs, chn[0].sttstcs, spc[i] );
    }
  }
  else if ( chn[0].thrdIndx > 0 )
  {
    InitFromLast ( chn );
  }

  hipEventRecord ( cdp[0].start, 0 );

  /* Run chain */
  printf ( ".................................................................\n" );
  printf ( " Start ...                                                  \n" );

  hiprandGenerateUniform ( cdp[0].curandGnrtr, chn[0].rndmVls, chn[0].nmbrOfRndmVls );

  int stpIndx = 0, sbstIndx;
  while ( stpIndx < chn[0].nmbrOfStps )
  {
    sbstIndx = 0;
    while ( sbstIndx < 2 )
    {
      Propose ( stpIndx, sbstIndx, chn );
      Priors ( mdl, chn );
      for ( int i = 0; i < NSPCTR; i++ )
      {
        ModelFluxes ( mdl, chn[0].nmbrOfWlkrs / 2, chn[0].prpsdWlkrs, i, spc[i] );
        FoldModel ( cdp, chn[0].nmbrOfWlkrs / 2, spc[i] );
        Stat ( chn[0].nmbrOfWlkrs / 2, spc[i] );
        SumUpStat ( cdp, 1, chn[0].nmbrOfWlkrs / 2, chn[0].prpsdSttstcs, spc[i] );
      }
      Update ( stpIndx, sbstIndx, chn );
      sbstIndx += 1;
    }
    ToChain ( stpIndx, chn );
    stpIndx += 1;
  }
  printf ( "      ... >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>> Done!\n" );

  hipEventRecord ( cdp[0].stop, 0 );
  hipEventSynchronize ( cdp[0].stop );
  hipEventElapsedTime ( &chn[0].elapsedTime, cdp[0].start, cdp[0].stop );

  hipEventRecord ( cdp[0].start, 0 );

  /* Autocorrelation function */
  int NN[RANK] = { chn[0].nmbrOfStps };
  cdp[0].cufftRes = hipfftPlanMany ( &cdp[0].cufftPlan, RANK, NN, NULL, 1, chn[0].nmbrOfStps, NULL, 1, chn[0].nmbrOfStps, HIPFFT_C2C, chn[0].nmbrOfWlkrs );
  if ( cdp[0].cufftRes != HIPFFT_SUCCESS ) { fprintf ( stderr, "CUFFT error: Direct Plan configuration failed" ); return 1; }
  ReturnChainFunction <<< Grid ( chn[0].nmbrOfWlkrs, chn[0].nmbrOfStps ), dimBlock >>> ( chn[0].nmbrOfStps, chn[0].nmbrOfWlkrs, 0, chn[0].chnOfWlkrs, chn[0].chnFnctn );
  AutocorrelationFunctionAveraged ( cdp[0].cufftRes, cdp[0].cublasStat, cdp[0].cublasHandle, cdp[0].cufftPlan, chn[0].nmbrOfStps, chn[0].nmbrOfWlkrs, chn[0].chnFnctn, chn[0].atCrrFnctn );

  hipEventRecord ( cdp[0].stop, 0 );
  hipEventSynchronize ( cdp[0].stop );
  hipEventElapsedTime ( &chn[0].cufftElapsedTime, cdp[0].start, cdp[0].stop );

  /* Autocorreation time */
  CumulativeSumOfAutocorrelationFunction ( chn[0].nmbrOfStps, chn[0].atCrrFnctn, chn[0].cmSmAtCrrFnctn );
  int MM = ChooseWindow ( chn[0].nmbrOfStps, 5e0f, chn[0].cmSmAtCrrFnctn );
  chn[0].atcTime = 2 * chn[0].cmSmAtCrrFnctn[MM] - 1e0f;
  printf ( ".................................................................\n" );
  printf ( " Autocorrelation time window -- %i\n", MM );
  printf ( " Autocorrelation time -- %.8E\n", chn[0].atcTime );
  printf ( " Autocorrelation time threshold -- %.8E\n", chn[0].nmbrOfStps / 5e1f );
  printf ( " Effective number of independent samples -- %.8E\n", chn[0].nmbrOfWlkrs * chn[0].nmbrOfStps / chn[0].atcTime );

  /* Elapsed time */
  printf ( ".................................................................\n" );
  printf ( " Time to generate: %3.1f ms\n", chn[0].elapsedTime );
  printf ( " Time to compute Autocorrelation Function: %3.1f ms\n", chn[0].cufftElapsedTime );
  printf ( "\n" );

  /* Write results to a file */
  SimpleWriteDataFloat ( "Autocor.out", chn[0].nmbrOfStps, chn[0].atCrrFnctn );
  SimpleWriteDataFloat ( "AutocorCM.out", chn[0].nmbrOfStps, chn[0].cmSmAtCrrFnctn );
  WriteChainToFile ( chn[0].thrdNm, chn[0].thrdIndx, chn[0].nmbrOfWlkrs, chn[0].nmbrOfStps, chn[0].chnOfWlkrs, chn[0].chnOfSttstcs );

  /* Destroy cuda related contexts and things: */
  DestroyAllTheCudaStaff ( cdp );

  /* Free memory: */
  FreeSpec ( spc );
  FreeChain ( chn );
  FreeModel ( mdl );

  // Reset the device and exit
  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application exits

  cdp[0].err = hipDeviceReset ( );
  if ( cdp[0].err != hipSuccess )
  {
    fprintf ( stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString ( cdp[0].err ) );
    exit ( EXIT_FAILURE );
  }

  return 0;
}

#endif // _GWMCMCCUDA_CU_
