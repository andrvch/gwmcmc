#ifndef _GWMCMCCUDA_CU_
#define _GWMCMCCUDA_CU_

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <fitsio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include "StrctrsAndFnctns.cuh"

int main ( int argc, char *argv[] ) {
  const int vrb = 1;

  Cupar cdp[1];
  cdp[0].dev = atoi ( argv[1] );

  initializeCuda ( cdp );

  if ( vrb ) {
    printf ( "\n" );
    printf ( ".................................................................\n" );
    printf ( " CUDA device ID: %d\n", cdp[0].dev );
    printf ( " CUDA device Name: %s\n", cdp[0].prop.name );
    printf ( " Driver API: v%d \n", cdp[0].driverVersion[0] );
    printf ( " Runtime API: v%d \n", cdp[0].runtimeVersion[0] );
  }

  Chain chn[1];
  const char *spcFl1 = argv[2];
  const char *spcFl2 = argv[3];
  const char *spcFl3 = argv[4];
  const char *spcFl4 = argv[5];
  const char *spcFl5 = argv[6];
  const char *spcFl6 = argv[7];
  const char *spcFl7 = argv[8];
  const char *spcFl8 = argv[9];
  const char *spcFl9 = argv[10];
  const char *spcFl10 = argv[11];
  const char *spcFl11 = argv[12];
  const char *spcFl12 = argv[13];
  const char *spcFl13 = argv[14];
  const char *spcFl14 = argv[15];
  const char *spcFl15 = argv[16];
  const char *spcFl16 = argv[17];
  const char *spcLst[NSPCTR11] = { spcFl1, spcFl2, spcFl3, spcFl4, spcFl5, spcFl6, spcFl7, spcFl8, spcFl9, spcFl10, spcFl11, spcFl12, spcFl13, spcFl14, spcFl15, spcFl16 };

  chn[0].name = argv[NSPCTR11+2];
  chn[0].nwl = atoi ( argv[NSPCTR11+3] );
  chn[0].nst = atoi ( argv[NSPCTR11+4] );
  chn[0].indx = atoi ( argv[NSPCTR11+5] );
  chn[0].dim = NPRS;
  chn[0].dlt = 1.E-4;

  Model mdl[1];
  Spectrum spc[NSPCTR];

  const float lwrNtcdEnrg1 = ( float ) atof ( argv[NSPCTR11+6] );
  const float hghrNtcdEnrg1 = ( float ) atof ( argv[NSPCTR11+7] );

  for ( int i = 0; i < NSPCTR; i++ ) {
    spc[i].lwrNtcdEnrg = lwrNtcdEnrg1;
    spc[i].hghrNtcdEnrg = hghrNtcdEnrg1;
  }

  InitializeModel ( mdl );

  SpecInfo ( spcLst, vrb, spc );
  SpecAlloc ( chn, spc );
  SpecData ( cdp, vrb, mdl, spc );

  spc[7].backscal_src = 1;
  spc[8].backscal_bkg = 1;

  allocateChain ( chn );

  chn[0].x0[0] = 5.9;
  chn[0].xbnd[0] = 5.5;
  chn[0].xbnd[1] = 6.5;

  chn[0].x0[1] = 0.0;
  chn[0].xbnd[2] = -25.;
  chn[0].xbnd[3] = 25.;

  chn[0].x0[2] = 0.0;
  chn[0].xbnd[4] = -25.;
  chn[0].xbnd[5] = 25.;

  chn[0].x0[3] = 0.0;
  chn[0].xbnd[6] = -25.;
  chn[0].xbnd[7] = 25.;

  chn[0].x0[4] = 1.5;
  chn[0].xbnd[8] = -25.;
  chn[0].xbnd[9] = 2.;

  chn[0].x0[5] = -5.;
  chn[0].xbnd[10] = -25.;
  chn[0].xbnd[11] = 25.;

  chn[0].x0[6] = 1.5;
  chn[0].xbnd[12] = -25.;
  chn[0].xbnd[13] = 25.;

  chn[0].x0[7] = -5.;
  chn[0].xbnd[14] = -25.;
  chn[0].xbnd[15] = 25.;

  chn[0].x0[8] = 1.5;
  chn[0].xbnd[16] = -25.;
  chn[0].xbnd[17] = 25.;

  chn[0].x0[9] = -5.;
  chn[0].xbnd[18] = -25.;
  chn[0].xbnd[19] = 25.;

  chn[0].x0[10] = 1.5;
  chn[0].xbnd[20] = -25.;
  chn[0].xbnd[21] = 25.;

  chn[0].x0[11] = -5.;
  chn[0].xbnd[22] = -25.;
  chn[0].xbnd[23] = 25.;

  chn[0].x0[12] = 1.5;
  chn[0].xbnd[24] = -25.;
  chn[0].xbnd[25] = 25.;

  chn[0].x0[13] = -5.;
  chn[0].xbnd[26] = -25.;
  chn[0].xbnd[27] = 25.;

  chn[0].x0[14] = 0.2;
  chn[0].xbnd[28] = 0.;
  chn[0].xbnd[29] = 25.;

  initializeChain ( cdp, chn, mdl, spc );

  /*
  hipDeviceSynchronize ();

  for ( int i = 0; i < spc[0].nmbrOfNtcdChnnls; i++ ) {
    printf ( " %2.2f ", spc[0].flddMdlFlxs[i]  );
    printf ( " %2.2f ", spc[0].chnnlSttstcs[i]  );
  }
  printf ( "\n" );

  for ( int i = 0; i < chn[0].nwl; i++ ) {
    printf ( " %2.2f ", chn[0].stt[i]  );
  }
  printf ( "\n" );
  */

  if ( vrb ) {
    printf ( ".................................................................\n" );
    printf ( " Start ...                                                  \n" );
  }

  hipEventRecord ( cdp[0].start, 0 );

  initializeRandomForStreach ( cdp, chn );

  chn[0].ist = 0;
  while ( chn[0].ist < chn[0].nst ) {
    chn[0].isb = 0;
    while ( chn[0].isb < 2 ) {
      streachMove ( cdp, chn );
      modelStatistic1 ( cdp, mdl, chn, spc );
      streachUpdate ( cdp, chn, mdl );
      chn[0].isb += 1;
    }
    saveCurrent ( chn );
    chn[0].ist += 1;
  }

  if ( vrb ) {
    printf ( "      ... >>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>> Done!\n" );
  }

  hipEventRecord ( cdp[0].stop, 0 );
  hipEventSynchronize ( cdp[0].stop );
  hipEventElapsedTime ( &chn[0].time, cdp[0].start, cdp[0].stop );

  if ( vrb ) {
    printf ( ".................................................................\n" );
    printf ( " Time to generate: %3.1f ms\n", chn[0].time );
    printf ( "\n" );
  }

  hipEventRecord ( cdp[0].start, 0 );

  averagedAutocorrelationFunction ( cdp, chn );

  hipEventRecord ( cdp[0].stop, 0 );
  hipEventSynchronize ( cdp[0].stop );
  hipEventElapsedTime ( &chn[0].time, cdp[0].start, cdp[0].stop );

  if ( vrb ) {
    printf ( ".................................................................\n" );
    printf ( " Autocorrelation time window -- %i\n", chn[0].mmm );
    printf ( " Autocorrelation time -- %.8E\n", chn[0].atcTime );
    printf ( " Autocorrelation time threshold -- %.8E\n", chn[0].nst / 5e1f );
    printf ( " Effective number of independent samples -- %.8E\n", chn[0].nwl * chn[0].nst / chn[0].atcTime );
    printf ( " Time to compute acor time: %3.1f ms\n", chn[0].time );
    printf ( "\n" );
  }

  /* Write results to a file */
  simpleWriteDataFloat ( "Autocor.out", chn[0].nst, chn[0].atcrrFnctn );
  simpleWriteDataFloat ( "AutocorCM.out", chn[0].nst, chn[0].cmSmAtCrrFnctn );
  writeChainToFile ( chn[0].name, chn[0].indx, chn[0].dim, chn[0].nwl, chn[0].nst, chn[0].smpls, chn[0].stat, chn[0].priors, chn[0].dist, chn[0].chiTwo );

  destroyCuda ( cdp );
  freeChain ( chn );
  FreeModel ( mdl );
  FreeSpec ( spc );

  // Reset the device and exit
  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application exits

  cdp[0].err = hipDeviceReset ();
  if ( cdp[0].err != hipSuccess ) {
    fprintf ( stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString ( cdp[0].err ) );
    exit ( EXIT_FAILURE );
  }

  return 0;
}

#endif // _GWMCMCCUDA_CU_
