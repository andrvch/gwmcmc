#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

__host__ void statisticsLoop ( const int n, const float *r, float *p ) {
  for ( int i = 0; i < n; i++ ) {
    p[i] = ( powf ( r[i], 2 ) + powf ( r[i+1], 2 ) < 1 ) * 1.;
  }
}

__host__ float statSum ( const int n, const float *p ) {
  float res = 0;
  for ( int i = 0; i < n; i++ ) {
    res += p[i];
  }
  return res;
}


int main ( int argc, char *argv[] ) {
  const int n = atoi ( argv[1] );

  hiprandGenerator_t gen;
  hipEvent_t start, stop;

  hipEventCreate ( &start );
  hipEventCreate ( &stop );
  hipEventRecord ( start, 0 );

  hiprandCreateGeneratorHost ( &gen, HIPRAND_RNG_PSEUDO_DEFAULT );
  hiprandSetPseudoRandomGeneratorSeed ( gen, 1234ULL );

  float *r, *p, res;
  hipMallocManaged ( ( void ** ) &r, 2 * n * sizeof ( float ) );
  hipMallocManaged ( ( void ** ) &p, n * sizeof ( float ) );

  hiprandGenerateUniform ( gen, r, 2 * n );
  statisticsLoop ( n, r, p );
  res = statSum ( n, p );

  float elapsedTime;
  hipEventRecord ( stop, 0 );
  hipEventSynchronize ( stop );
  hipEventElapsedTime ( &elapsedTime, start, stop );

  printf ( "%1.8f\n", 4 * res / n );
  printf ( " Time to generate: %3.1f ms\n", elapsedTime );


  hiprandDestroyGenerator ( gen );

  hipFree ( r );
  hipFree ( p );

  return 0;
}
